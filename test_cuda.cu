#include "hip/hip_runtime.h"
#include "wukong.h"
#include <gtest/gtest.h>
#include <cstdlib>
#include <ctime>

static inline void assert_array_eq(const float *a, const float *b, size_t n)
{
    for (int i = 0; i < n; i++)
        EXPECT_NEAR(a[i], b[i], 1e-5);
}

class cudaEnv : public ::testing::Environment {
public:
  void SetUp() override { cuda_init(); }
  void TearDown() override { cuda_fini(); }
};

int main(int argc, char* argv[]) {
    ::testing::InitGoogleTest(&argc, argv);
    ::testing::AddGlobalTestEnvironment(new cudaEnv);
    return RUN_ALL_TESTS();
}

TEST(Cuda, cuda_matmul)
{
    int b = 2;
    int r = 2;
    int c = 3;
    int oc = 4;

    float out[b * r * oc] = {0};
    float inp[b * r * c] = {0.680375f, -0.211234f, 0.566198f,
                            0.596880f, 0.823295f, -0.604897f,
                            -0.329554f, 0.536459f, -0.444451f,
                            0.107940f, -0.045206f, 0.257742f};
    float weight[c * oc] = {-0.270431f, 0.026802f, 0.904459f,
                            0.832390f, 0.271423f, 0.434594f,
                            -0.716795f, 0.213938f, -0.967399f,
                            -0.514226f, -0.725537f, 0.608353f};
    float bias[oc] = {0.1f, 0.2f, 0.3f, 0.4f};
    float res[b * r * oc] = {0.422447f, 0.955070f, -0.780620f, 0.547840f,
                             -0.586453f, 0.657414f, 0.633470f, -0.872253f,
                             -0.198488f, -0.121866f, 1.080953f, -0.090139f,
                             0.302715f, 0.389591f, -0.036381f, 0.534091f};
    float res_nob[] = {0.322447f, 0.755070f, -1.080620f, 0.147840f,
                       -0.686453f, 0.457414f, 0.333470f, -1.272253f,
                       -0.298488f, -0.321866f, 0.780953f, -0.490139f,
                       0.202715f, 0.189591f, -0.336381f, 0.134091f};
    void *d_out = cuda_malloc(b * r * oc * sizeof(float));
    void *d_inp = cuda_malloc(b * r * c * sizeof(float));
    void *d_weight = cuda_malloc(c * oc * sizeof(float));
    void *d_bias = cuda_malloc(oc * sizeof(float));
    cuda_to_device(d_inp, inp, b * r * c * sizeof(float));
    cuda_to_device(d_weight, weight, c * oc * sizeof(float));
    cuda_to_device(d_bias, bias, oc * sizeof(float));

    cuda_matmul(d_out, d_inp, d_weight, d_bias, b * r, c, oc);
    cuda_to_host(out, d_out, b * r * oc * sizeof(float));
    assert_array_eq(res, out, b * r * oc);
    cuda_matmul(d_out, d_inp, d_weight, nullptr, b * r, c, oc);
    cuda_to_host(out, d_out, b * r * oc * sizeof(float));
    assert_array_eq(res_nob, out, b * r * oc);

    cuda_free(d_out);
    cuda_free(d_inp);
    cuda_free(d_weight);
    cuda_free(d_bias);
}

TEST(Cuda, cuda_softmax)
{
    float inp[2 * 3] = {2.0f, 2.0f, 2.0f, 4.0f, 1000.0f, 1.0f};
    float out[2 * 3] = {0};
    float res[] = {0.333333f, 0.333333f, 0.333333f, 0.000000f, 1.000000f, 0.000000f};

    void *d_out = cuda_malloc(2 * 3 * sizeof(float));
    void *d_inp = cuda_malloc(2 * 3 * sizeof(float));
    cuda_to_device(d_inp, inp, 2 * 3 * sizeof(float));
    cuda_softmax(d_out, d_inp, 2, 3);
    cuda_to_host(out, d_out, 2 * 3 * sizeof(float));
    assert_array_eq(res, out, 6);
    cuda_free(d_out);
    cuda_free(d_inp);
}

TEST(Cuda, cuda_mha_attention)
{
    int batch = 2;
    int row = 4;
    int NH = 1;
    int HS = 2;
    int col = NH * HS;

    float inp[batch * row * col * 3] = {
        0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, // Batch1
        0.7f, 0.8f, 0.9f, 1.0f, 1.1f, 1.2f,
        1.3f, 1.4f, 1.5f, 1.6f, 1.7f, 1.8f,
        1.9f, 2.0f, 2.1f, 2.2f, 2.3f, 2.4f,
        0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, // Batch2
        0.7f, 0.8f, 0.9f, 1.0f, 1.1f, 1.2f,
        1.3f, 1.4f, 1.5f, 1.6f, 1.7f, 1.8f,
        1.9f, 2.0f, 2.1f, 2.2f, 2.3f, 2.4f,
    };

    float out[batch * row * col] = {0};
    float res[batch * row * col] = {
        0.500000f, 0.600000f,
        0.892363f, 0.992363f,
        1.479998f, 1.579998f,
        2.161403f, 2.261404f,
        0.500000f, 0.600000f,
        0.892363f, 0.992363f,
        1.479998f, 1.579998f,
        2.161403f, 2.261404f,
    };

    void *d_out = cuda_malloc(batch * row * col * sizeof(float));
    void *d_inp = cuda_malloc(batch * row * col * 3 * sizeof(float));
    cuda_to_device(d_inp, inp, batch * row * col * 3 * sizeof(float));

    cuda_mha_attention(d_out, d_inp, batch, row, NH, HS);
    cuda_to_host(out, d_out, batch * row * col * sizeof(float));
    assert_array_eq(res, out, batch * row * col);
    // printm(out, batch, row, col);

    cuda_free(d_out);
    cuda_free(d_inp);
}

TEST(Cuda, cuda_gqa_attention)
{
    { // Case: Single element input
        int batch = 1;
        int row = 1;
        int qNH = 1;
        int kvNH = 1;
        int HS = 1;

        float inp[3] = {0.1f, 0.2f, 0.3f};
        float out[1] = {0};

        float res[1] = {0.3f};

        void *d_out = cuda_malloc(1 * sizeof(float));
        void *d_inp = cuda_malloc(3 * sizeof(float));
        cuda_to_device(d_inp, inp, 3 * sizeof(float));

        cuda_gqa_attention(d_out, d_inp, batch, row, qNH, kvNH, HS);
        cuda_to_host(out, d_out, 1 * sizeof(float));
        assert_array_eq(res, out, 1);

        cuda_free(d_out);
        cuda_free(d_inp);
    }

    {
        int batch = 2;
        int row = 2;
        int qNH = 4;
        int kvNH = 2;
        int HS = 2;
        int qSize = qNH * HS;
        int kvSize = kvNH * HS;

        float inp[batch * row * (qSize + 2*kvSize)] = {
            // Batch 1, Row 1
            0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f,     // Q (4 heads * 2 dims)
            0.1f, 0.2f, 0.3f, 0.4f,                             // K (2 heads * 2 dims)
            0.5f, 0.6f, 0.7f, 0.8f,                             // V (2 heads * 2 dims)
            // Batch 1, Row 2
            1.1f, 1.2f, 1.3f, 1.4f, 1.5f, 1.6f, 1.7f, 1.8f,     // Q
            0.9f, 1.0f, 1.1f, 1.2f,                             // K
            1.3f, 1.4f, 1.5f, 1.6f,                             // V
            // Batch 2, Row 1 (same pattern as batch 1)
            0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f,
            0.1f, 0.2f, 0.3f, 0.4f,
            0.5f, 0.6f, 0.7f, 0.8f,
            // Batch 2, Row 2
            1.1f, 1.2f, 1.3f, 1.4f, 1.5f, 1.6f, 1.7f, 1.8f,
            0.9f, 1.0f, 1.1f, 1.2f,
            1.3f, 1.4f, 1.5f, 1.6f
        };

        float out[batch * row * qSize] = {0};

        float res[batch * row * qSize] = {
            // Batch 0
            0.500000f, 0.600000f, 0.700000f, 0.800000f, 0.500000f, 0.600000f, 0.700000f, 0.800000f,
            1.128813f, 1.228813f, 1.357295f, 1.457295f, 1.181927f, 1.281927f, 1.402936f, 1.502936f,
            // Batch 1
            0.500000f, 0.600000f, 0.700000f, 0.800000f, 0.500000f, 0.600000f, 0.700000f, 0.800000f,
            1.128813f, 1.228813f, 1.357295f, 1.457295f, 1.181927f, 1.281927f, 1.402936f, 1.502936f
        };

        void *d_out = cuda_malloc(batch * row * qSize * sizeof(float));
        void *d_inp = cuda_malloc(batch * row * (qSize + 2*kvSize) * sizeof(float));
        cuda_to_device(d_inp, inp, batch * row * (qSize + 2*kvSize) * sizeof(float));

        cuda_gqa_attention(d_out, d_inp, batch, row, qNH, kvNH, HS);
        cuda_to_host(out, d_out, batch * row * qSize * sizeof(float));
        assert_array_eq(res, out, batch * row * qSize);

        cuda_free(d_out);
        cuda_free(d_inp);
    }
}

TEST(Cuda, cuda_mqa_attention)
{
    int batch = 2;
    int row = 2;
    int qNH = 4;
    int HS = 2;
    int qSize = qNH * HS;
    int kvSize = HS; // kvNH = 1 for MQA

    float inp[batch * row * (qSize + 2*kvSize)] = {
        // Batch 1, Row 1
        0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f,     // Q (4 heads * 2 dims)
        0.1f, 0.2f,                                          // K (1 head * 2 dims)
        0.5f, 0.6f,                                          // V (1 head * 2 dims)
        // Batch 1, Row 2
        1.1f, 1.2f, 1.3f, 1.4f, 1.5f, 1.6f, 1.7f, 1.8f,     // Q
        0.9f, 1.0f,                                          // K
        1.3f, 1.4f,                                          // V
        // Batch 2, Row 1 (same pattern as batch 1)
        0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f,
        0.1f, 0.2f,
        0.5f, 0.6f,
        // Batch 2, Row 2
        1.1f, 1.2f, 1.3f, 1.4f, 1.5f, 1.6f, 1.7f, 1.8f,
        0.9f, 1.0f,
        1.3f, 1.4f
    };

    float out[batch * row * qSize] = {0};

    float res[batch * row * qSize] = {
        // Batch 0
        0.500000f, 0.600000f, 0.500000f, 0.600000f, 0.500000f, 0.600000f, 0.500000f, 0.600000f,
        1.128813f, 1.228813f, 1.157295f, 1.257295f, 1.181927f, 1.281927f, 1.202936f, 1.302936f,
        // Batch 1
        0.500000f, 0.600000f, 0.500000f, 0.600000f, 0.500000f, 0.600000f, 0.500000f, 0.600000f,
        1.128813f, 1.228813f, 1.157295f, 1.257295f, 1.181927f, 1.281927f, 1.202936f, 1.302936f
    };

    void *d_out = cuda_malloc(batch * row * qSize * sizeof(float));
    void *d_inp = cuda_malloc(batch * row * (qSize + 2*kvSize) * sizeof(float));
    cuda_to_device(d_inp, inp, batch * row * (qSize + 2*kvSize) * sizeof(float));

    cuda_mqa_attention(d_out, d_inp, batch, row, qNH, HS);
    cuda_to_host(out, d_out, batch * row * qSize * sizeof(float));
    assert_array_eq(res, out, batch * row * qSize);

    cuda_free(d_out);
    cuda_free(d_inp);
}

TEST(Cuda, cuda_rmsnorm)
{
    int batch = 2;
    int row = 2;
    int col = 4;

    float inp[batch * row * col] = {
        0.1f, 0.2f, 0.3f, 0.4f,
        0.5f, 0.6f, 0.7f, 0.8f,
        0.9f, 1.0f, 1.1f, 1.2f,
        1.3f, 1.4f, 1.5f, 1.6f
    };

    float weight[col] = {0.5f, 0.6f, 0.7f, 0.8f};

    float out[batch * row * col] = {0};

    float res[batch * row * col] = {
        0.182562f, 0.438149f, 0.766761f, 1.168397f,
        0.379045f, 0.545824f, 0.742928f, 0.970354f,
        0.426160f, 0.568214f, 0.729208f, 0.909142f,
        0.446948f, 0.577595f, 0.721993f, 0.880144f,
    };

    void *d_out = cuda_malloc(batch * row * col * sizeof(float));
    void *d_inp = cuda_malloc(batch * row * col * sizeof(float));
    void *d_weight = cuda_malloc(col * sizeof(float));

    cuda_to_device(d_inp, inp, batch * row * col * sizeof(float));
    cuda_to_device(d_weight, weight, col * sizeof(float));

    cuda_rmsnorm(d_out, d_inp, d_weight, batch * row, col, 1e-5);
    cuda_to_host(out, d_out, batch * row * col * sizeof(float));

    assert_array_eq(res, out, batch * row * col);

    cuda_free(d_out);
    cuda_free(d_inp);
    cuda_free(d_weight);
}

TEST(Cuda, cuda_swiglu)
{
    int batch = 2;
    int row = 2;
    int col = 3;
    int hidden_size = 4;

    float inp[batch * row * col] = {
        0.1f, 0.2f, 0.3f,
        0.4f, 0.5f, 0.6f,
        0.1f, 0.2f, 0.3f,
        0.4f, 0.5f, 0.6f
    };

    // Expected output tensor
    float res[batch * row * hidden_size] = {
        0.010485f, 0.059323f, 0.155615f, 0.306913f,
        0.059323f, 0.405260f, 1.149139f, 2.347071f,
        0.010485f, 0.059323f, 0.155615f, 0.306913f,
        0.059323f, 0.405260f, 1.149139f, 2.347071f
    };

    // concatenate the weights
    float weights_fc[2 * hidden_size * col] = {
        0.1f, 0.2f, 0.3f,
        0.4f, 0.5f, 0.6f,
        0.7f, 0.8f, 0.9f,
        1.0f, 1.1f, 1.2f,
        0.1f, 0.2f, 0.3f,
        0.4f, 0.5f, 0.6f,
        0.7f, 0.8f, 0.9f,
        1.0f, 1.1f, 1.2f
    };
    void *d_out = cuda_malloc(batch * row * hidden_size * sizeof(float));
    void *d_inp = cuda_malloc(batch * row * col * sizeof(float));
    void *d_fcout = cuda_malloc(batch * row * 2 * hidden_size * sizeof(float));
    void *d_weights_fc = cuda_malloc(2 * hidden_size * col * sizeof(float));

    cuda_to_device(d_inp, inp, batch * row * col * sizeof(float));
    cuda_to_device(d_weights_fc, weights_fc, 2 * hidden_size * col * sizeof(float));

    cuda_matmul(d_fcout, d_inp, d_weights_fc, nullptr, batch * row, col, 2 * hidden_size);
    cuda_swiglu(d_out, d_fcout, batch, row, hidden_size);

    float out[batch * row * hidden_size] = {0};
    cuda_to_host(out, d_out, batch * row * hidden_size * sizeof(float));

    assert_array_eq(res, out, batch * row * hidden_size);

    cuda_free(d_out);
    cuda_free(d_inp);
    cuda_free(d_fcout);
    cuda_free(d_weights_fc);
}

// Precompute the freqs of the RoPE rotation for the given HS(HeadSize) and theta
// return array of size HS/2
void get_freqs(floatX *freqs, int HS, float theta)
{
    // helper function that (on the CPU!) precomputes the freqs_cis for the RoPE rotation
    // same as precompute_freqs_cis_real in rope.py
    for (int i = 0; i < HS / 2; i++) {

        // calculate the frequency for the (i, i+1)th dimension
        float freq = 1.0f / powf(theta, (float)(2 * i) / HS);
        const int scale_factor = 8;
        const int low_freq_factor = 1;
        const int high_freq_factor = 4;
        const int old_context_len = 8192;  // original llama3 length
        const float low_freq_wavelen = (float)old_context_len / low_freq_factor;
        const float high_freq_wavelen = (float)old_context_len / high_freq_factor;
        float wavelen = 2.0f * M_PI / freq;
        if (wavelen < high_freq_wavelen) {
            // skip; keep freq as is
        }
        else if (wavelen > low_freq_wavelen) {
            // scale down by scale_factor
            freq /= scale_factor;
        }
        else {
            // smooth transition between scaled and unscaled
            float smooth = ((float)old_context_len / wavelen - low_freq_factor) / (high_freq_factor - low_freq_factor);
            freq = (1.0f - smooth) * freq / scale_factor + smooth * freq;
        }
        freqs[i] = freq;
    }
}

// return array of size row * HS
void get_freqs_cis(floatX *freqs_cis, int dim, int end, float theta, int use_scaled)
{
    // helper function that (on the CPU!) precomputes the freqs_cis for the RoPE rotation
    // same as precompute_freqs_cis_real in rope.py
    for (int i = 0; i < dim / 2; i++) {

        // calculate the frequency for the (i, i+1)th dimension
        float freq = 1.0f / powf(theta, (float)(2 * i) / dim);
        if (use_scaled) {
            const int scale_factor = 8;
            const int low_freq_factor = 1;
            const int high_freq_factor = 4;
            const int old_context_len = 8192;  // original llama3 length
            const float low_freq_wavelen = (float)old_context_len / low_freq_factor;
            const float high_freq_wavelen = (float)old_context_len / high_freq_factor;
            float wavelen = 2.0f * M_PI / freq;
            if (wavelen < high_freq_wavelen) {
                // skip; keep freq as is
            } else if (wavelen > low_freq_wavelen) {
                // scale down by scale_factor
                freq /= scale_factor;
            } else {
                // smooth transition between scaled and unscaled
                float smooth = ((float)old_context_len / wavelen - low_freq_factor) / (high_freq_factor - low_freq_factor);
                freq = (1.0f - smooth) * freq / scale_factor + smooth * freq;
            }
        }

        // iterate over all time steps, calculate the angle, and store the cos/sin
        for (int t = 0; t < end; t++) {
            float angle = (float)t * freq;
            freqs_cis[t * dim + 2 * i] = cosf(angle);     // real part
            freqs_cis[t * dim + 2 * i + 1] = sinf(angle); // imaginary part
        }
    }
}

TEST(Cuda, cuda_rope)
{
    int batch = 2;
    int row = 3;
    int NH = 2;
    int HS = 2;

    float qkv[batch * row * NH * HS] = {
        // batch 0
        0.1f, 0.2f, 0.3f, 0.4f,
        0.5f, 0.6f, 0.7f, 0.8f,
        0.9f, 1.0f, 1.1f, 1.2f,
        // batch 1
        1.3f, 1.4f, 1.5f, 1.6f,
        1.7f, 1.8f, 1.9f, 2.0f,
        2.1f, 2.2f, 2.3f, 2.4f,
    };
    float fc[HS/2] = {0};
    float fc_res[HS/2] = {
        1.000000f,
    };
    float out[batch * row * NH * HS] = {0};
    float res[batch * row * NH * HS] = {
        0.100000f, 0.200000f, 0.30000f, 0.400000f,
        -2.347315e-01, 7.449169e-01, -2.949652e-01, 1.021272e+00,
        -1.283830e+00, 4.022209e-01, -1.548918e+00, 5.008510e-01,
        1.300000e+00, 1.400000e+00, 1.500000e+00, 1.600000e+00,
        -0.596134f, 2.403045f, -0.656368f, 2.679399f,
        -2.874363e+00, 9.940016e-01, -3.139452e+00, 1.092632e+00
    };

    get_freqs(fc, HS, 10000.0f);
    assert_array_eq(fc_res, fc, HS / 2);

    void *d_qkv = cuda_malloc(batch * row * NH * HS * sizeof(float));
    void *d_fc = cuda_malloc(HS / 2 * sizeof(float));

    cuda_to_device(d_qkv, qkv, batch * row * NH * HS * sizeof(float));
    cuda_to_device(d_fc, fc, HS / 2 * sizeof(float));
    cuda_rope(d_qkv, d_qkv, d_fc, batch, row, NH, HS); // update qkv in-place
    cuda_to_host(out, d_qkv, batch * row * NH * HS * sizeof(float));
    assert_array_eq(res, out, batch * row * NH * HS);

    cuda_free(d_qkv);
    cuda_free(d_fc);

    NH = 1;
    HS = 4;
    float fc_res2[HS/2] = {
        1.000000f, 0.010000f,
    };
    float res2[batch * row * 3 * NH * HS] = {
        0.100000f, 0.200000f, 0.300000f, 0.400000f,
        -2.347315e-01, 7.449169e-01, 6.919651e-01, 8.069599e-01,
        -1.283830e+00, 4.022209e-01, 1.075782e+00, 1.221759e+00,
        1.300000e+00, 1.400000e+00, 1.500000e+00, 1.600000e+00,
        -5.961339e-01, 2.403045e+00, 1.879905e+00, 2.018900e+00,
        -2.874363e+00, 9.940016e-01, 2.251543e+00, 2.445517e+00,
    };
    get_freqs(fc, HS, 10000.0f);
    assert_array_eq(fc_res2, fc, HS / 2);
    d_qkv = cuda_malloc(batch * row * NH * HS * sizeof(float));
    d_fc = cuda_malloc(HS / 2 * sizeof(float));
    cuda_to_device(d_qkv, qkv, batch * row * NH * HS * sizeof(float));
    cuda_to_device(d_fc, fc, HS / 2 * sizeof(float));
    cuda_rope(d_qkv, d_qkv, d_fc, batch, row, NH, HS); // update qkv in-place
    cuda_to_host(out, d_qkv, batch * row * NH * HS * sizeof(float));
    assert_array_eq(res2, out, batch * row * NH * HS);

    cuda_free(d_qkv);
    cuda_free(d_fc);
}

TEST(Cuda, cuda_embedding) {
    int batch = 2;
    int row = 4;
    int col = 4;
    int vocab_size = 6;

    int inp[] = {0, 2, 1, 3, 4, 5, 1, 0};
    float embd[] = {
        0.1f, 0.2f, 0.3f, 0.4f,
        0.4f, 0.5f, 0.6f, 0.7f,
        0.7f, 0.8f, 0.9f, 1.0f,
        1.0f, 1.1f, 1.2f, 1.3f,
        1.1f, 1.2f, 1.3f, 1.4f,
        1.4f, 1.5f, 1.6f, 1.7f,
    };
    float h_out[batch * row * col];

    int *d_inp;
    float *d_embd, *d_out;

    hipMalloc(&d_inp, sizeof(int) * batch * row);
    hipMalloc(&d_embd, sizeof(float) * vocab_size * col);
    hipMalloc(&d_out, sizeof(float) * batch * row * col);

    hipMemcpy(d_inp, inp, sizeof(int) * batch * row, hipMemcpyHostToDevice);
    hipMemcpy(d_embd, embd, sizeof(float) * vocab_size * col, hipMemcpyHostToDevice);

    cuda_embedding(d_out, d_inp, d_embd, batch, row, col);

    hipMemcpy(h_out, d_out, sizeof(float) * batch * row * col, hipMemcpyDeviceToHost);

    float expected[] = {
        0.1f, 0.2f, 0.3f, 0.4f,
        0.7f, 0.8f, 0.9f, 1.0f,
        0.4f, 0.5f, 0.6f, 0.7f,
        1.0f, 1.1f, 1.2f, 1.3f,
        1.1f, 1.2f, 1.3f, 1.4f,
        1.4f, 1.5f, 1.6f, 1.7f,
        0.4f, 0.5f, 0.6f, 0.7f,
        0.1f, 0.2f, 0.3f, 0.4f,
    };
    assert_array_eq(h_out, expected, batch * row * col);

    hipFree(d_inp);
    hipFree(d_embd);
    hipFree(d_out);
}
TEST(Cuda, cuda_cat) {
    int arow = 2;
    int brow = 1;
    int col = 3;

    float a[arow * col] = {1.0f, 2.0f, 3.0f,
                           4.0f, 5.0f, 6.0f};
    float b[brow * col] = {7.0f, 8.0f, 9.0f};
    float out[(arow + brow) * col] = {0};

    float expected[(arow + brow) * col] = {
        1.0f, 2.0f, 3.0f,
        4.0f, 5.0f, 6.0f,
        7.0f, 8.0f, 9.0f,
    };

    void *d_out = cuda_malloc((arow + brow) * col * sizeof(float));
    void *d_a = cuda_malloc(arow * col * sizeof(float));
    void *d_b = cuda_malloc(brow * col * sizeof(float));

    cuda_to_device(d_a, a, arow * col * sizeof(float));
    cuda_to_device(d_b, b, brow * col * sizeof(float));

    cuda_cat(d_out, d_a, d_b, arow, brow, col);
    cuda_to_host(out, d_out, (arow + brow) * col * sizeof(float));

    assert_array_eq(out, expected, (arow + brow) * col);

    cuda_free(d_out);
    cuda_free(d_a);
    cuda_free(d_b);
}

TEST(Cuda, cuda_div)
{
    int row = 2;
    int col = 3;

    float a[row * col] = {6.0f, 12.0f, 18.0f,
                          24.0f, 30.0f, 36.0f};
    float b[row * col] = {2.0f, 3.0f, 6.0f,
                          8.0f, 5.0f, 9.0f};
    float out[row * col] = {0};

    float expected[row * col] = {
        3.0f, 4.0f, 3.0f,
        3.0f, 6.0f, 4.0f,
    };

    void *d_out = cuda_malloc(row * col * sizeof(float));
    void *d_a = cuda_malloc(row * col * sizeof(float));
    void *d_b = cuda_malloc(row * col * sizeof(float));

    cuda_to_device(d_a, a, row * col * sizeof(float));
    cuda_to_device(d_b, b, row * col * sizeof(float));

    cuda_div(d_out, d_a, d_b, row, col);
    cuda_to_host(out, d_out, row * col * sizeof(float));

    assert_array_eq(out, expected, row * col);

    cuda_free(d_out);
    cuda_free(d_a);
    cuda_free(d_b);
}

TEST(Cuda, cuda_dequantize)
{
    int row = 2;
    int nb = 2;
    int col = nb * 32;
    int type = GGML_TYPE_Q8_0;

    uint8_t inp[row * nb * 34] = {
        8, 52, 4, 8, 12, 16, 20, 24, 28, 32, 36, 40, 44, 48, 52, 56, 60, 64, 67, 71, 75, 79, 83, 87, 91, 95, 99, 103, 107, 111, 115, 119, 123, 127,
        0, 25, 0, 252, 8, 244, 16, 236, 25, 227, 33, 219, 41, 211, 49, 203, 57, 195, 66, 186, 74, 178, 82, 170, 90, 162, 98, 154, 107, 145, 115, 137, 123, 129,
        0, 12, 0, 4, 8, 12, 16, 20, 25, 29, 33, 37, 41, 45, 49, 53, 57, 61, 66, 70, 74, 78, 82, 86, 90, 94, 98, 102, 107, 111, 115, 119, 123, 127,
        154, 1, 0, 4, 8, 12, 16, 20, 25, 29, 33, 37, 41, 45, 49, 53, 57, 61, 66, 70, 74, 78, 82, 86, 90, 94, 98, 102, 107, 111, 115, 119, 123, 127,
    };

    float out[row * col] = {0};

    float expected[row * col] = {
        1.007812e+00, 2.015625e+00, 3.023438e+00, 4.031250e+00, 5.039062e+00, 6.046875e+00, 7.054688e+00, 8.062500e+00, 9.070312e+00, 1.007812e+01, 1.108594e+01, 1.209375e+01, 1.310156e+01, 1.410938e+01, 1.511719e+01, 1.612500e+01, 1.688086e+01, 1.788867e+01, 1.889648e+01, 1.990430e+01, 2.091211e+01, 2.191992e+01, 2.292773e+01, 2.393555e+01, 2.494336e+01, 2.595117e+01, 2.695898e+01, 2.796680e+01, 2.897461e+01, 2.998242e+01, 3.099023e+01, 3.199805e+01,
        0.000000e+00, -9.765625e-03, 1.953125e-02, -2.929688e-02, 3.906250e-02, -4.882812e-02, 6.103516e-02, -7.080078e-02, 8.056641e-02, -9.033203e-02, 1.000977e-01, -1.098633e-01, 1.196289e-01, -1.293945e-01, 1.391602e-01, -1.489258e-01, 1.611328e-01, -1.708984e-01, 1.806641e-01, -1.904297e-01, 2.001953e-01, -2.099609e-01, 2.197266e-01, -2.294922e-01, 2.392578e-01, -2.490234e-01, 2.612305e-01, -2.709961e-01, 2.807617e-01, -2.905273e-01, 3.002930e-01, -3.100586e-01,
        0.000000e+00, 9.765625e-04, 1.953125e-03, 2.929688e-03, 3.906250e-03, 4.882812e-03, 6.103516e-03, 7.080078e-03, 8.056641e-03, 9.033203e-03, 1.000977e-02, 1.098633e-02, 1.196289e-02, 1.293945e-02, 1.391602e-02, 1.489258e-02, 1.611328e-02, 1.708984e-02, 1.806641e-02, 1.904297e-02, 2.001953e-02, 2.099609e-02, 2.197266e-02, 2.294922e-02, 2.392578e-02, 2.490234e-02, 2.612305e-02, 2.709961e-02, 2.807617e-02, 2.905273e-02, 3.002930e-02, 3.100586e-02,
        0.000000e+00, 9.775162e-05, 1.955032e-04, 2.932549e-04, 3.910065e-04, 4.887581e-04, 6.109476e-04, 7.086992e-04, 8.064508e-04, 9.042025e-04, 1.001954e-03, 1.099706e-03, 1.197457e-03, 1.295209e-03, 1.392961e-03, 1.490712e-03, 1.612902e-03, 1.710653e-03, 1.808405e-03, 1.906157e-03, 2.003908e-03, 2.101660e-03, 2.199411e-03, 2.297163e-03, 2.394915e-03, 2.492666e-03, 2.614856e-03, 2.712607e-03, 2.810359e-03, 2.908111e-03, 3.005862e-03, 3.103614e-03
    };

    void *d_out = cuda_malloc(row * col * sizeof(float));
    void *d_inp = cuda_malloc(row * 2 * sizeof(block_q8_0));

    cuda_to_device(d_inp, inp, row * 2 * sizeof(block_q8_0));

    cuda_dequantize(d_out, d_inp, row, col, type);
    cuda_to_host(out, d_out, row * col * sizeof(float));
    assert_array_eq(expected, out, row * col);

    cuda_free(d_out);
    cuda_free(d_inp);
}