#include "hip/hip_runtime.h"
#include "wukong.h"

#define QK_K 256

dtype_info dtype_infos[GGML_TYPE_COUNT] = {
    {"F32", 1, 4},
    {"F16", 1, 2},
    {"GGML_TYPE_Q4_0", 32, 2 + 16},
    {"GGML_TYPE_Q4_1", 32, 2 + 2 + 16},
    {"GGML_TYPE_Q4_2", 32, 2 + 2 + 16},
    {"GGML_TYPE_Q4_3", 32, 2 + 2 + 16},
    {"GGML_TYPE_Q5_0", 32, 2 + 4 + 16},
    {"GGML_TYPE_Q5_1", 32, 2 + 2 + 4 + 16},
    {"GGML_TYPE_Q8_0", 32, 2 + 32},
    {"GGML_TYPE_Q8_1", 32, 4 + 4 + 32},
    {"GGML_TYPE_Q2_K", 256, 2 + 2 + QK_K / 16 + QK_K / 4},
    {"GGML_TYPE_Q3_K", 256, 2 + QK_K / 4 + QK_K / 8 + 12},
    {"GGML_TYPE_Q4_K", 256, 2 + 2 + QK_K / 2 + 12},
    {"GGML_TYPE_Q5_K", 256, 2 + 2 + QK_K / 2 + QK_K / 8 + 12},
    {"GGML_TYPE_Q6_K", 256, 2 + QK_K / 2 + QK_K / 4 + QK_K / 16},
    {"GGML_TYPE_Q8_K", 256, 4 + QK_K + QK_K / 8},
    {"GGML_TYPE_IQ2_XXS", 256, 2 + QK_K / 4},
    {"GGML_TYPE_IQ2_XS", 256, 2 + QK_K / 4 + QK_K / 32},
    {"GGML_TYPE_IQ3_XXS", 256, 2 + QK_K / 4 + QK_K / 8},
    {"GGML_TYPE_IQ1_S", 256, 2 + QK_K / 8 + QK_K / 16},
    {"GGML_TYPE_IQ4_NL", 32, 2 + 16},
    {"GGML_TYPE_IQ3_S", 256, 2 + QK_K / 4 + QK_K / 8 + QK_K / 32 + 4},
    {"GGML_TYPE_IQ2_S", 256, 2 + QK_K / 4 + QK_K / 16},
    {"GGML_TYPE_IQ4_XS", 256, 2 + 2 + QK_K / 2 + QK_K / 64},
    {"Int8", 1, 1},
    {"Int16", 1, 2},
    {"Int32", 1, 4},
    {"Int64", 1, 8},
    {"F64", 1, 8},
    {"GGML_TYPE_IQ1_M", 256, QK_K / 8 + QK_K / 16 + QK_K / 32},
    {"BF16", 1, 2},
    {"GGML_TYPE_Q4_0_4_4", 32, 2 + 16},
    {"GGML_TYPE_Q4_0_4_8", 32, 2 + 16},
    {"GGML_TYPE_Q4_0_8_8", 32, 2 + 16},
    {"GGML_TYPE_TQ1_0", 256, 2 + 4 * 13},
    {"GGML_TYPE_TQ2_0", 256, 2 + 64}
};

// cuBLAS workspace. Only Hopper needs 32 MB, for others 4 is OK
static size_t cublaslt_workspace_size = 4 * 1024 * 1024;
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;
static hipblasLtHandle_t cublaslt_handle;
static hipblasHandle_t cublas_handle;
static hipStream_t main_stream;
static int deviceIdx = 0;
__attribute_maybe_unused__ static int cuda_arch_major = 0;
__attribute_maybe_unused__ static int cuda_arch_minor = 0;
__attribute_maybe_unused__ static int cuda_num_SMs = 0; // for persistent threads where we want 1 threadblock per SM
__attribute_maybe_unused__ static int cuda_threads_per_SM = 0;    // needed to calculate how many blocks to launch to fill up the GPU
__attribute_maybe_unused__ static int cuda_threads_per_block = 0;
__attribute_maybe_unused__ static int cuda_warp_size = 0; // warp size of the GPU
__attribute_maybe_unused__ static int cuda_max_shared_mem_per_block = 0;

__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__device__ float warp_reduce_max(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

__device__ __forceinline__ void warp_reduce_max(float& val, int& idx) {
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        float other_val = __shfl_down_sync(0xffffffff, val, offset);
        int other_idx = __shfl_down_sync(0xffffffff, idx, offset);
        if (other_val > val) {
            val = other_val;
            idx = other_idx;
        }
    }
}

// Handles both scaling of attention scores and softmax computation with causal masking
// inp/out shape: (B, NH, T, T)
__global__ void scaled_softmax_kernel(float* out, const float* inp, int B, int NH, int T, float scale)
 {
    extern __shared__ float shared[];
    int batch_idx = blockIdx.x / (NH * T); // batch index
    int head_idx = (blockIdx.x / T) % NH;  // head index
    int row_idx = blockIdx.x % T;          // row index within the attention matrix
    int tid = threadIdx.x;
    int warpId = threadIdx.x / WARP_SIZE;         // warp index within a block
    int laneId = threadIdx.x % WARP_SIZE;         // thread index within a warp
    int warpsPerBlock = blockDim.x / WARP_SIZE;

    // shared memory layout: first half for max values, second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // calculate base index for this thread block's row
    int row_start = (batch_idx * NH * T * T) + (head_idx * T * T) + (row_idx * T);
    const float* x = inp + row_start;

    // Step 1: Find maximum while applying scale and causal mask
    float maxval = -INFINITY;
    for (int i = tid; i < T; i += blockDim.x) {
        float val = (i <= row_idx) ? x[i] * scale : -INFINITY;
        maxval = fmaxf(maxval, val);
    }

    // warp-level reduction for maxval
    maxval = warp_reduce_max(maxval);

    // write per-warp maxval to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // final reduction for maxval across warps
    if (tid == 0) {
        float val = maxvals[0];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        maxvals[0] = val;
    }
    __syncthreads();

    // broadcast max to all threads
    float offset = maxvals[0];

    // Step 2: Compute exp(x - max) while respecting causal mask
    float sumval = 0.0f;
    for (int i = tid; i < T; i += blockDim.x) {
        float val = (i <= row_idx) ? expf(x[i] * scale - offset) : 0.0f;
        out[row_start + i] = val;  // store intermediate result
        sumval += val;
    }

    // warp-level reduction for sum
    sumval = warpReduceSum(sumval);

    // write per-warp sum to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // final reduction for sum across warps
    if (tid == 0) {
        float val = sumvals[0];
        for (int i = 1; i < warpsPerBlock; i++) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();

    // Step 3: Normalize by sum
    float sum = sumvals[0];
    float inv_sum = 1.0f / sum;

    // write final normalized values
    for (int i = tid; i < T; i += blockDim.x) {
        if (i <= row_idx) {
            out[row_start + i] *= inv_sum;
        } else {
            out[row_start + i] = 0.0f;
        }
    }
}

__global__ void softmax_kernel(float* output, const float* input, int row, int col) {
    extern __shared__ float shared_mem[];
    float* row_max = shared_mem;                    // First part of shared memory for max values
    float* row_sum = &shared_mem[blockDim.x / WARP_SIZE];  // Second part for sum values

    int tid = threadIdx.x;
    int lane_id = tid % WARP_SIZE;
    int warp_id = tid / WARP_SIZE;
    int warps_per_block = blockDim.x / WARP_SIZE;
    int row_idx = blockIdx.x;

    if (row_idx >= row) return;

    // Step 1: Find maximum value in the row
    float thread_max = -INFINITY;
    for (int i = tid; i < col; i += blockDim.x) {
        thread_max = fmaxf(thread_max, input[row_idx * col + i]);
    }

    // Warp-level reduction for max
    thread_max = warp_reduce_max(thread_max);

    // Store per-warp results
    if (lane_id == 0) {
        row_max[warp_id] = thread_max;
    }
    __syncthreads();

    // Final reduction for max across warps
    if (tid == 0) {
        float max_val = row_max[0];
        for (int i = 1; i < warps_per_block; i++) {
            max_val = fmaxf(max_val, row_max[i]);
        }
        row_max[0] = max_val;
    }
    __syncthreads();

    // Step 2: Compute exp(x - max) and sum
    float max_val = row_max[0];
    float thread_sum = 0.0f;

    for (int i = tid; i < col; i += blockDim.x) {
        float val = expf(input[row_idx * col + i] - max_val);
        output[row_idx * col + i] = val;  // Store intermediate result
        thread_sum += val;
    }

    // Warp-level reduction for sum
    thread_sum = warpReduceSum(thread_sum);

    // Store per-warp sums
    if (lane_id == 0) {
        row_sum[warp_id] = thread_sum;
    }
    __syncthreads();

    // Final reduction for sum across warps
    if (tid == 0) {
        float sum = row_sum[0];
        for (int i = 1; i < warps_per_block; i++) {
            sum += row_sum[i];
        }
        row_sum[0] = sum;
    }
    __syncthreads();

    // Step 3: Normalize by sum
    float inv_sum = 1.0f / row_sum[0];
    for (int i = tid; i < col; i += blockDim.x) {
        output[row_idx * col + i] *= inv_sum;
    }
}

__global__ void unpermute_kernel(float *out, const float * inp, int B, int N, int NH, int d)
{
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = inp[idx];
    }
}

__global__ void permute_kernel(float* q, float* k, float* v,
                               const float* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * NH * N * d) {
        return;
    }

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]
    int b = idx / (NH * N * d);
    int rest = idx % (NH * N * d);
    int nh_ = rest / (N * d);
    rest = rest % (N * d);
    int n = rest / d;
    int d_ = rest % d;
    int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
    q[idx] = __ldcs(&inp[inp_idx]);
    k[idx] = __ldcs(&inp[inp_idx + NH * d]);
    v[idx] = __ldcs(&inp[inp_idx + 2 * (NH * d)]);
}

__global__ void add_bias_kernel(float* out, const float* bias, int T, int OC)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < T * OC; i += stride) {
        int col = i % OC;
        out[i] += bias[col];
    }
}

__global__ void swiglu_kernel(float* out, const float* inp, int B, int T, int C)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * T * C) {
        int b = idx / (T * C);
        int t = (idx / C) % T;
        int c = idx % C;

        int fc1_idx = (b * T * 2 * C) + (t * 2 * C) + c;
        int fc2_idx = fc1_idx + C;

        float swish_val = inp[fc2_idx] / (1.0f + expf(-inp[fc2_idx]));
        out[idx] = swish_val * inp[fc1_idx];
    }
}

template <typename T>
__global__ void rope_qkv_kernel(T *out, const T *inp, const float *freqs,
                               int batch, int row, int NH, int kvNH, int HS)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int HS_half = HS / 2;
    int total_heads = NH + kvNH;
    int total = batch * row * total_heads * HS_half;

    if (idx >= total)
        return;

    int b = idx / (row * total_heads * HS_half);
    int r = (idx / (total_heads * HS_half)) % row;
    int h = (idx / HS_half) % total_heads;
    int d = idx % HS_half;

    int freq_idx = r * HS + 2 * d;
    float c = freqs[freq_idx];
    float s = freqs[freq_idx + 1];

    int base = b * (row * (NH + 2 * kvNH) * HS) + r * ((NH + 2 * kvNH) * HS) + h * HS + 2 * d;
    if constexpr (std::is_same<T, float>::value) {
        float x_real = inp[base];
        float x_imag = inp[base + 1];

        float result_real = x_real * c - x_imag * s;
        float result_imag = x_real * s + x_imag * c;

        out[base] = result_real;
        out[base + 1] = result_imag;
    } else if constexpr (std::is_same<T, hip_bfloat16>::value) {
	    float x_real = bf16_to_f32(inp[base]);
	    float x_imag = bf16_to_f32(inp[base + 1]);

	    float result_real = x_real * c - x_imag * s;
	    float result_imag = x_real * s + x_imag * c;
	    out[base] = f32_to_bf16(result_real);
	    out[base + 1] = f32_to_bf16(result_imag);
    } else {
        panic("Unsupported type for rope_qkv_kernel");
    }
}

__global__ void rope_kernel(float *out, const float *inp, const float *freqs, int B, int T, int NH, int HS)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int HS_half = HS / 2;
    if (idx >= B * T * NH * HS_half)
        return;

    // decode the individual indices
    int b = idx / (T * NH * HS_half);
    int t = (idx / (NH * HS_half)) % T;
    int h = (idx / HS_half) % NH;
    int d = idx % HS_half;
    int idx_bt = b * (T * NH * HS) + t * (NH * HS);
    int idx_bth = idx_bt + h * HS;
    int idxi = idx_bth + 2 * d; // index in the input

    // fetch and compute frequency
    float freq = freqs[d];
    float angle = t * freq;
    float freqs_cos = cosf(angle);
    float freqs_sin = sinf(angle);

    // fetch the input
    float x_real = inp[idxi];
    float x_imag = inp[idxi + 1];
    // apply the rotation
    out[idxi] = x_real * freqs_cos - x_imag * freqs_sin;
    out[idxi + 1] = x_real * freqs_sin + x_imag * freqs_cos;
}

__global__ void get_embeddings_kernel(void* out, const int* inp, const void* embd, int batch, int row, size_t bytes_per_row)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch * row)
        return;

    int b = idx / row;
    int t = idx % row;
    int ix = inp[b * row + t];

    char* dst = (char*)out + (b * row + t) * bytes_per_row;
    const char* src = (const char*)embd + ix * bytes_per_row;

    memcpy(dst, src, bytes_per_row);
}

void cuda_matmul_cublas(float *out, const float *inp, const float *weight, const float *bias,
                        int row, int column, int oc)
{
    const float alpha = 1.0f;
    const float beta = 0.0f;
    // cublas sees us transposed, so we want out(oc, row) = weight(oc, c) @ inp(c, row) + bias
    cublas_check(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, oc, row, column, /* M, N, K*/
                            &alpha, weight, oc, inp, column, &beta, out, oc));
    if (bias != NULL) {
        int block_size = cuda_threads_per_block;
        int grid_size = CEIL_DIV(oc * row, block_size);
        add_bias_kernel<<<grid_size, block_size, 0, main_stream>>>(out, bias, row, oc);
        cuda_check(hipGetLastError());
    }
}

template<typename T>
void cuda_matmul_cublaslt(T *out, const T *inp, const T *weight, const T *bias,
                        int row, int column, int oc)
{
    int res;
    bool has_bias = (bias != nullptr);
    hipblasLtMatmulDesc_t desc;
    hipblasLtMatmulPreference_t pref;
    hipblasLtMatrixLayout_t inp_layout, weight_layout, out_layout, bias_layout;
    hipblasLtMatmulHeuristicResult_t heuristic;
    hipblasOperation_t notrans = HIPBLAS_OP_N;
    hipblasOperation_t trans = HIPBLAS_OP_T;
    hipblasLtEpilogue_t epilogue = has_bias ? HIPBLASLT_EPILOGUE_BIAS : HIPBLASLT_EPILOGUE_DEFAULT;
    hipDataType data_type;
    hipblasComputeType_t compute_type;

    if constexpr (std::is_same<T, float>::value) {
        data_type = HIP_R_32F;
        compute_type = cublas_compute_type;
    } else if constexpr (std::is_same<T, hip_bfloat16>::value) {
        data_type = HIP_R_16BF;
        compute_type = HIPBLAS_COMPUTE_32F;
        // Forces any reductions during matrix multiplications to use the compute type and not the output type
        hipblasSetMathMode(cublas_handle, (hipblasMath_t)(HIPBLAS_DEFAULT_MATH | HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION));
    } else {
        panic("Unsupported type for cuda_matmul_cublaslt");
    }

    /*
     * Cuda is colum-major, for row-major Array, if we want to get: out = inp @ weight.T, 'out' should be 'out.T'.
     * Mathematically, out.T = weight @ inp.T. Since cuda is colum-major, 'weight' should be weight.T, 'inp.T' should be inp.
     * so calculating out.T = weight.T & inp.
     */
    cublas_check(hipblasLtMatmulDescCreate(&desc, compute_type, HIP_R_32F));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(trans)));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSB, &notrans, sizeof(notrans)));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    cublas_check(hipblasLtMatrixLayoutCreate(&weight_layout, data_type, column, oc, column));
    cublas_check(hipblasLtMatrixLayoutCreate(&inp_layout, data_type, column, row, column));
    cublas_check(hipblasLtMatrixLayoutCreate(&out_layout, data_type, oc, row, oc));
    cublas_check(hipblasLtMatrixLayoutCreate(&bias_layout, data_type, oc, 1, oc));

    if (has_bias && (uintptr_t)bias % 16 != 0)
        panic("bias must be aligned to 16 bytes");

    cublas_check(hipblasLtMatmulPreferenceCreate(&pref));
    cublas_check(hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    cublas_check(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, desc, weight_layout, inp_layout, out_layout,
                out_layout, pref, 1, &heuristic, &res));
    if (res == 0)
        panic("No algorithm found: row=%d, column=%d, oc=%d, has_bias=%d", row, column, oc, has_bias);

    const float alpha = 1.0f, beta = 0.0f;
    cublas_check(hipblasLtMatmul(cublaslt_handle, desc, &alpha, weight, weight_layout, inp, inp_layout, &beta,
                out, out_layout, out, out_layout, &heuristic.algo, cublaslt_workspace, cublaslt_workspace_size, main_stream));

    cublas_check(hipblasLtMatmulPreferenceDestroy(pref));
    cublas_check(hipblasLtMatmulDescDestroy(desc));
    cublas_check(hipblasLtMatrixLayoutDestroy(weight_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(inp_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(out_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(bias_layout));
}

void cuda_matmul_cublaslt_f32(void *out, const void *inp, const void *weight, const void *bias,
                        int row, int column, int oc)
{
    cuda_matmul_cublaslt<float>(
        static_cast<float*>(out),
        static_cast<const float*>(inp),
        static_cast<const float*>(weight),
        static_cast<const float*>(bias),
        row, column, oc
    );
}

void cuda_matmul_cublaslt_bf16(void *out, const void *inp, const void *weight, const void *bias,
                        int row, int column, int oc)
{
    cuda_matmul_cublaslt<hip_bfloat16>(
        static_cast<hip_bfloat16*>(out),
        static_cast<const hip_bfloat16*>(inp),
        static_cast<const hip_bfloat16*>(weight),
        static_cast<const hip_bfloat16*>(bias),
        row, column, oc
    );
}

__global__ void div_kernel(float *out, const float *a, const float *b, int row, int col)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = row * col;

    if (idx < total_elements) {
        out[idx] = a[idx] / b[idx];
    }
}

__global__ void dequantize_Q8_0(float *out, const block_q8_0 *inp, int row, int nb, int bs)
{
    extern __shared__ block_q8_0 shared_block[];
    int block_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_blocks = row * nb;

    if (block_idx >= total_blocks)
        return;

    int r = block_idx / nb; // row index
    int b = block_idx % nb; // block index

    const block_q8_0 *block = inp + r * nb + b;
    shared_block[threadIdx.x] = *block;
    __syncthreads();

    float scale = __half2float(shared_block[threadIdx.x].scale);
    #pragma unroll
    for (int i = 0; i < bs; ++i) {
	    int out_idx = r * nb * bs + b * bs + i;
	    out[out_idx] = scale * shared_block[threadIdx.x].d[i];
    }
}

__global__ void add_kernel(float* out, const float* a, const float* b, int row, int col)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = row * col;

    if (idx >= size)
        return;

    float4 *out4 = (float4 *)out;
    const float4 *a4 = (const float4 *)a;
    const float4 *b4 = (const float4 *)b;

    float4 va = a4[idx];
    float4 vb = b4[idx];
    float4 vout;
    vout.x = va.x + vb.x;
    vout.y = va.y + vb.y;
    vout.z = va.z + vb.z;
    vout.w = va.w + vb.w;
    out4[idx] = vout;
}

template <typename T>
__global__ void repeat_qkv_kernel(T* __restrict__ replicated_qkv, const T* __restrict__ gqa_qkv,
                               int B, int N, int NH, int HD, int replicate_factor) {
    // we have a single tensor gqa_qkv of shape (B, N, (NH + 2*(NH/replicate_factor)) * HD)
    // we want to replicate it into (B, N, 3 * NH * HD)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * N * 3 * NH * HD) { return; }
    int idx_flat = idx; // keep backup

    // decode the output index
    int d = idx % HD;
    idx /= HD;
    int nh = idx % NH;
    idx /= NH;
    int c = idx % 3;
    idx /= 3;
    int n = idx % N;
    int b = idx / N;

    int inp_idx;
    int nh_total = NH + 2 * (NH / replicate_factor);
    if (c == 0) {
        inp_idx = b * N * nh_total * HD + n * nh_total * HD + 0 * NH * HD + nh * HD + d;
    } else if (c == 1) {
        inp_idx = b * N * nh_total * HD + n * nh_total * HD + 1 * NH * HD + (nh / replicate_factor) * HD + d;
    } else {
        inp_idx = b * N * nh_total * HD + n * nh_total * HD + (NH * HD + (NH / replicate_factor) * HD) + (nh / replicate_factor) * HD + d;
    }

    replicated_qkv[idx_flat] = __ldcs(&gqa_qkv[inp_idx]);
}

__global__ void get_row_kernel(float *out, const float *inp, int batch, int row, int col, int idx)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b < batch) {
        const float *src = inp + b * row * col + idx * col;
        float *dst = out + b * col;
        memcpy(dst, src, col * sizeof(float));
    }
}

__global__ void argmax_kernel(int *out, const float *inp, int row, int col)
{
    __shared__ float smax[WARP_SIZE];  // Max values per warp
    __shared__ int sidx[WARP_SIZE];    // Corresponding indices

    int r = blockIdx.x;                // One row per block
    int tid = threadIdx.x;             // Thread ID
    int wid = tid / WARP_SIZE;         // Warp ID
    int lane = tid % WARP_SIZE;        // Lane within warp

    if (r >= row) return;

    // Each thread's running max
    float max_val = -INFINITY;
    int max_idx = -1;

    for (int i = tid; i < col; i += blockDim.x) {
        float val = inp[r * col + i];
        if (val > max_val) {
            max_val = val;
            max_idx = i;
        }
    }

    // Warp-level reduction
    warp_reduce_max(max_val, max_idx);

    // Write warp results to shared memory
    if (lane == 0) {
        smax[wid] = max_val;
        sidx[wid] = max_idx;
    }
    __syncthreads();

    // Final reduction across warps by first warp
    if (wid == 0) {
        max_val = (lane < blockDim.x/WARP_SIZE) ? smax[lane] : -INFINITY;
        max_idx = (lane < blockDim.x/WARP_SIZE) ? sidx[lane] : -1;

        warp_reduce_max(max_val, max_idx);

        if (lane == 0) {
            out[r] = max_idx;
        }
    }
}

template <typename T>
__global__ void rmsnorm_kernel(T* __restrict__ out, const float* __restrict__ inp,
                              const float* __restrict__ weight, int N, int C, float eps)
{
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> warp = cg::tiled_partition<WARP_SIZE>(block);

    __shared__ float shared_sum2[WARP_SIZE]; // One element per warp for squared sum

    int num_warps = blockDim.x / WARP_SIZE;
    int warp_id = threadIdx.x / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int idx = blockIdx.x; // One block per row

    // Point to current sequence position
    const float* x = inp + idx * C;

    // Thread coarsening through the row
    float thread_sum2 = 0.0f;

    // Each thread accumulates multiple elements
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float xi = x[i];
        thread_sum2 += xi * xi;
    }

    // Warp-level reduction for sum of squares
    float warp_sum2 = cg::reduce(warp, thread_sum2, cg::plus<float>{});

    // Store warp-level results to shared memory
    if (lane_id == 0) {
        shared_sum2[warp_id] = warp_sum2;
    }
    __syncthreads();

    // Load results from shared memory to threads, pad with zeros for out-of-bounds threads
    warp_sum2 = (lane_id < num_warps) ? shared_sum2[lane_id] : 0.0f;

    // Reduce the warp-level results
    float block_sum2 = cg::reduce(warp, warp_sum2, cg::plus<float>{});

    block_sum2 /= C; // mean(x**2)
    float s = rsqrtf(block_sum2 + eps); // 1 / sqrt(mean(x**2) + eps)

    // Apply normalization and scaling
    T* o = out + idx * C;
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float val = __ldcs(x + i);
        float normalized = val * s * weight[i]; // x / sqrt(mean(x**2) + eps) * weight
        if constexpr (std::is_same<T, float>::value) {
            __stcs(o + i, normalized);
        } else {
            __stcs(o + i, f32_to_bf16(normalized));
        }
    }
}

template <typename T>
void cuda_rmsnorm(T *out, const float *inp, const float *weight, int row, int col, float eps)
{
    const int block_size = 256;
    rmsnorm_kernel<T><<<row, block_size, 0, main_stream>>>(out, inp, weight, row, col, eps);
    cuda_check(hipGetLastError());
}

extern "C" {
void cuda_init(int idx)
{
    srand(0);   // determinism

    deviceIdx = idx;

    cuda_check(hipSetDevice(deviceIdx));
    cuda_check(hipStreamCreate(&main_stream));

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    cuda_num_SMs = deviceProp.multiProcessorCount;
    cuda_threads_per_SM = deviceProp.maxThreadsPerMultiProcessor;
    cuda_arch_major = deviceProp.major;
    cuda_arch_minor = deviceProp.minor;
    cuda_threads_per_block = deviceProp.maxThreadsPerBlock;
    cuda_warp_size = deviceProp.warpSize;
    cuda_max_shared_mem_per_block = deviceProp.sharedMemPerBlock;
    // printf("CUDA device: %s, major %d, minor %d, num_SMs: %d, threads_per_SM: %d, threads_per_block: %d, warp_size: %d\n",
    //        deviceProp.name, cuda_arch_major, cuda_arch_minor, cuda_num_SMs, cuda_threads_per_SM, cuda_threads_per_block, cuda_warp_size);
    printf("Running on GPU %d: %s, Compute Capability %d.%d\n", deviceIdx, deviceProp.name, cuda_arch_major, cuda_arch_minor);

    cublas_check(hipblasCreate(&cublas_handle));
    cublas_check(hipblasLtCreate(&cublaslt_handle));
    cublaslt_workspace = cuda_malloc(cublaslt_workspace_size);

    // Set the stream for cublas handle
    cublas_check(hipblasSetStream(cublas_handle, main_stream));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = cuda_arch_major >= 8 ? 1 : 0;
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
}

void cuda_fini(void)
{
    cuda_free(cublaslt_workspace);
    cublas_check(hipblasDestroy(cublas_handle));
    cublas_check(hipblasLtDestroy(cublaslt_handle));
    cuda_check(hipStreamDestroy(main_stream));
}


void* cuda_malloc(size_t size)
{
    void *ptr;
    cuda_check(hipMallocAsync(&ptr, size, main_stream));
    return ptr;
}

void cuda_free(void* ptr)
{
    cuda_check(hipFreeAsync(ptr, main_stream));
}

void cuda_to_device(void* dst, void* src, size_t size)
{
    cuda_check(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, main_stream));
}

void cuda_to_host(void* dst, void* src, size_t size)
{

    cuda_check(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, main_stream));
    cuda_check(hipStreamSynchronize(main_stream));
}

/*
 * Fused matrix multiplication with optional bias addition: out = inp @ weight^T + bias
 *
 * @param out: output matrix(row, oc)
 * @param inp: input matrix(row, column)
 * @param weight: weight matrix(oc, column)
 * @param bias: optional bias vector(oc) (can be NULL)
 * @param row: input row size
 * @param column: input column size
 * @param oc: output column size
 */
void cuda_matmul(void *out, const void *inp, const void *weight, const void *bias,
                int row, int column, int oc, int dtype)
{
    if (dtype == GGML_TYPE_BF16) {
        return cuda_matmul_cublaslt_bf16(out, inp, weight, bias, row, column, oc);
    } else if (dtype == GGML_TYPE_F32) {
        return cuda_matmul_cublaslt_f32(out, inp, weight, bias, row, column, oc);
    } else {
        void *dw = cuda_malloc(oc * column * sizeof(float));
        cuda_dequantize(dw, weight, oc, column, dtype);
        cuda_matmul_cublaslt_f32(out, inp, dw, bias, row, column, oc);
        cuda_free(dw);
        return;
    }
}

/*
 * Row-wise cuda_softmax
 * @param output: shape (row, column)
 * @param input: shape (row, column)
 * @param row: row size
 * @param col: column size
 */
void cuda_softmax(void* output, void* input, int row, int col)
{
    const int block_size = 256;
    const int shared_mem_size = (2 * (block_size / WARP_SIZE)) * sizeof(float); // Space for max and sum values
    softmax_kernel<<<row, block_size, shared_mem_size, main_stream>>>((float *)output, (const float *)input, row, col);
    cuda_check(hipGetLastError());
}

/*
 * GQA scaled dot product attention
 *
 * @param out: output matrix(batch, row, col) where col = qNH * HS
 * @param inp: input matrix(batch, row, (qNH + 2 * kvNH) * HS) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param qNH: number of Q heads
 * @param kvNH: number of K and V heads
 * @param HS: head size
 */
void cuda_gq_sdpa(void *out, const void *inp, int batch, int row, int qNH, int kvNH, int HS)
{
    void *qkv = cuda_malloc(batch * row * 3 * qNH * HS * sizeof(float));
    cuda_repeat_qkv(qkv, inp, batch, row, qNH, kvNH, HS);
    cuda_mh_sdpa(out, qkv, batch, row, qNH, HS);
    cuda_free(qkv);
}

/* Root Mean Square Layer Normalization: x / sqrt(mean(x^2) + eps) * weight
 *
 * @param out: output matrix(row, col)
 * @param inp: input matrix(row, col)
 * @param weight: weight matrix(col)
 * @param row: row size
 * @param col: column size
 * @param eps: epsilon value
 */
void cuda_rmsnorm_f32(void *out, const void *inp, const void *weight, int row, int col, float eps)
{
    cuda_rmsnorm<float>((float *)out, (const float *)inp, (const float *)weight, row, col, eps);
}

void cuda_rmsnorm_bf16(void *out, const void *inp, const void *weight, int row, int col, float eps)
{
    cuda_rmsnorm<hip_bfloat16>((hip_bfloat16 *)out, (const float *)inp, (const float *)weight, row, col, eps);
}

// swiglu: y = swish(fc2(x)) * fc1(x), where swish(x) = x / (1 + exp(-x)), fc1 and fc2 are fully connected layers
// @param out: output matrix(batch, row, col)
// @param inp: input matrix(batch, row, 2*col), concatenated fc1 and fc2 outputs along the last dimension
void cuda_swiglu(void *out, const void *inp, int batch, int row, int col)
{
    int block_size = 256;
    int grid_size = CEIL_DIV(batch * row * col, block_size);
    swiglu_kernel<<<grid_size, block_size, 0, main_stream>>>((float *)out, (const float *)inp, batch, row, col);
    cuda_check(hipGetLastError());
}

/*
 * Vanilla multi-head scaled dot product attention
 *
 * attention = softmax(Q@K^T/sqrt(HS)) @ V
 *
 * @param out: output matrix(batch, row, col)
 * @param inp: input matrix(batch, row, 3 * col) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param NH: number of heads
 * @param HS: head size
 * @attention col = NH * HS
 */
void cuda_mh_sdpa(void *out, const void *inp, int batch, int row, int NH, int HS)
{
    float *qkv, *att, *vatt;

    // Allocate space for broadcasted K and V
    size_t q_size = (batch * NH * row * HS) * sizeof(float);
    size_t qkv_size = 3 * q_size;
    size_t att_size = batch * NH * row * row * sizeof(float);

    qkv = (float *)cuda_malloc(qkv_size);
    // try best to reuse input buffer
    vatt = (float *)inp;
    att = (float *)cuda_malloc(att_size);

    float *q = qkv;
    float *k = qkv + batch * NH * row * HS;
    float *v = k + batch * NH * row * HS;

    // Permute input
    // q: (batch, row, NH, HS) -> (batch, NH, row, HS)
    // k: (batch, row, NH, HS) -> (batch, NH, row, HS)
    // v: (batch, row, NH, HS) -> (batch, NH, row, HS)
    //
    // Tradeoff: it uses more memory for the broadcasted K and V tensors, but this should be acceptable
    // given the benefits in simplicity and performance improvements (remove for-loop in following matmul)
    int total_threads = batch * NH * row * HS;
    int block_size = 256;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size, 0, main_stream>>>(q, k, v, (const float*)inp, batch, row, NH, HS);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Batched matrix multiplication: Q @ K^T
    cublas_check(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                            row, row, HS,
                            &alpha,
                            k, HS, row * HS,
                            q, HS, row * HS,
                            &beta,
                            att, row, row * row,
                            batch * NH));

    // Apply scaled softmax with causal masking
    float scale = 1.0f / sqrtf(HS);
    int softmax_block_size = 256;
    size_t shared_mem_size = 2 * (softmax_block_size / 32) * sizeof(float);
    int grid_size = batch * NH * row;
    scaled_softmax_kernel<<<grid_size, softmax_block_size, shared_mem_size, main_stream>>>(
        att, att, batch, NH, row, scale);

    // Batched matrix multiplication: attention @ V
    cublas_check(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            HS, row, row,
                            &alpha,
                            v, HS, row * HS,
                            att, row, row * row,
                            &beta,
                            vatt, HS, row * HS,
                            batch * NH));

    // Unpermute result from (batch, NH, row, HS) -> (batch, row, NH, HS)
    num_blocks = CEIL_DIV(batch * row * NH * HS, block_size);
    unpermute_kernel<<<num_blocks, block_size, 0, main_stream>>>((float *)out, vatt, batch, row, NH, HS);

    cuda_free(qkv);
    cuda_free(att);
}

/*
 * Multi query scaled dot product attention
 *
 * @param out: output matrix(batch, row, col) where col = qNH * HS
 * @param inp: input matrix(batch, row, (qNH + 2 * kvNH) * HS) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param qNH: number of Q heads
 * @param HS: head size
 */
void cuda_mq_sdpa(void *out, const void *inp, int batch, int row, int qNH, int HS)
{
    void *qkv = cuda_malloc(batch * row * 3 * qNH * HS * sizeof(float));
    cuda_repeat_qkv(qkv, inp, batch, row, qNH, 1, HS);
    cuda_mh_sdpa(out, qkv, batch, row, qNH, HS);
    cuda_free(qkv);
}

/*
 * RoPE: Rotated Positional Embedding
 *
 * @param out: output matrix(batch, row, NH + 2*kvNH, HS) where NH is for Q and kvNH each for K,V
 * @param inp: input matrix(batch, row, NH + 2*kvNH, HS) q, k, v concatenated along the last dimension
 * @freqs_cis: cos and sin frequencies for each element in q, k
 * @param batch: batch size
 * @param row: row size
 * @param NH: number of query heads
 * @param kvNH: number of key/value heads
 * @param HS: head size
 */
void cuda_rope_qkv(void *out, const void *inp, const void *freqs, int batch, int row, int NH, int kvNH, int HS)
{
    int block_size = 256;
    // We only need threads for Q and K sections, V will be untouched
    int total_threads = batch * row * (NH + kvNH) * HS / 2;
    int num_blocks = CEIL_DIV(total_threads, block_size);

    rope_qkv_kernel<float><<<num_blocks, block_size, 0, main_stream>>>((float *)out, (const float *)inp, (const float *)freqs,
                                               batch, row, NH, kvNH, HS);
    cuda_check(hipGetLastError());
}

/*
 * RoPE: Rotated Positional Embedding for a single tensor
 *
 * @param out: output matrix(batch, row, NH, HS)
 * @param inp: input matrix(batch, row, NH, HS)
 * @freqs: raw frequency tensor to compute the rotation angle (HS/2)
 * @param batch: batch size
 * @param row: row size
 * @param NH: number of heads
 * @param HS: head size
 */
void cuda_rope(void *out, const void *inp, const void *freqs, int batch, int row, int NH, int HS)
{
    int block_size = 256;
    int total_threads = batch * row * NH * HS / 2;  // divided by 2 since we process pairs
    int num_blocks = CEIL_DIV(total_threads, block_size);
    rope_kernel<<<num_blocks, block_size, 0, main_stream>>>((float *)out, (const float *)inp, (const float *)freqs, batch, row, NH, HS);
    cuda_check(hipGetLastError());
}

/*
 * Get the embeddings for the given indices using the embedding table
 *
 * @param out: output matrix(batch, row, col)
 * @param inp: input matrix(batch, row)
 * @param embd: embedding table (vacob_size, col)
 * @param batch: batch size
 * @param row: row size (number of indices)
 * @param col: column size (embedding size)
 */
void cuda_embedding(void* out, const void *inp, const void *embd, int batch, int row, int col, int dtype)
{
    if (dtype < 0 || dtype >= GGML_TYPE_COUNT)
        panic("Unsupported quantization type: %d", dtype);

    auto info = dtype_infos[dtype];
    assert(col % info.block_size == 0);
    size_t bytes_per_row = col / info.block_size * info.type_size;

    const int block_size = 256;
    const int N = batch * row;  // One thread per row
    const int grid_size = CEIL_DIV(N, block_size);

    if (dtype == GGML_TYPE_F32 || dtype == GGML_TYPE_BF16) {
        get_embeddings_kernel<<<grid_size, block_size, 0, main_stream>>>(out, (const int*)inp, embd, batch, row, bytes_per_row);
        cuda_check(hipGetLastError());
        return;
    }
    void *dout = cuda_malloc(batch * row * bytes_per_row);
    get_embeddings_kernel<<<grid_size, block_size, 0, main_stream>>>(dout, (const int*)inp, embd, batch, row, bytes_per_row);
    cuda_dequantize(out, dout, batch * row, col, dtype);
    cuda_check(hipGetLastError());
    cuda_free(dout);
}

/*
 * Concatenate the input tensors along the first dimension
 *
 * @param out: output matrix(arow + brow, col)
 * @param a: input matrix(arow, col)
 * @param b: input matrix(brow, col)
 * @param arow: row size of a
 * @param brow: row size of b
 * @param col: column size
 */
void cuda_cat(void *out, const void *a, const void *b, int arow, int brow, int col, int dtype)
{
    auto info = dtype_infos[dtype];
    size_t asize = arow * col * info.type_size / info.block_size;
    size_t bsize = brow * col * info.type_size / info.block_size;

    cuda_check(hipMemcpyAsync(out, a, asize, hipMemcpyDeviceToDevice, main_stream));
    cuda_check(hipMemcpyAsync((char *)out + asize, b, bsize, hipMemcpyDeviceToDevice, main_stream));
}

/*
 * Element-wise division a / b
 *
 * @param out: output matrix(row, col)
 * @param a: input matrix(row, col)
 * @param b: input matrix(row, col)
 * @param row: row size
 * @param col: column size
 */
void cuda_div(void *out, const void *a, const void *b, int row, int col)
{
    int block_size = 256;
    int total_threads = row * col;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    div_kernel<<<num_blocks, block_size, 0, main_stream>>>((float *)out, (const float *)a, (const float *)b, row, col);
    cuda_check(hipGetLastError());
}

/*
 * Dequantize the quantized input tensor from dtype to float
 *
 * @param out: output matrix(row, col)
 * @param inp: input matrix(row, col)
 * @param row: row size
 * @param col: column size
 * @param type: quantization dtype
 */
void cuda_dequantize(void *out, const void *inp, int row, int col, int type)
{
    if (type < 0 || type >= GGML_TYPE_COUNT)
        panic("Unsupported quantization type: %d", type);

    auto info = dtype_infos[type];
    int nb = col / info.block_size;
    int bs = info.block_size;
    int total_blocks = row * nb;
    int block_size = 256;
    int num_blocks = CEIL_DIV(total_blocks, block_size);
    size_t shared_mem_size = block_size * sizeof(block_q8_0);
    assert(shared_mem_size <= cuda_max_shared_mem_per_block);
    switch (type) {
    case GGML_TYPE_Q8_0:
	    dequantize_Q8_0<<<num_blocks, block_size, shared_mem_size, main_stream>>>((float *)out, (const block_q8_0 *)inp, row, nb, bs);
	    break;
    default:
	    panic("Unsupported quantization type: %s", dtype_infos[type].name);
	}
    cuda_check(hipGetLastError());
}

/*
 * Element-wise addition out = a + b
 *
 * @param out: output matrix(row, col)
 * @param a: input matrix(row, col)
 * @param b: input matrix(row, col)
 * @param row: row size
 * @param col: column size
 */
void cuda_add(void* out, const void* a, const void* b, int row, int col)
{
    const int total_size = row * col;
    const int block_size = 256;
    // Each thread handles 4 elements when using float4
    const int grid_size = CEIL_DIV(total_size, block_size * 4);

    assert(col % 4 == 0);
    add_kernel<<<grid_size, block_size, 0, main_stream>>>((float*)out, (const float*)a, (const float*)b, row, col);
    cuda_check(hipGetLastError());
}

void cuda_group_query_attention(void *out, const void *embeds, const void *freqs, const void *norm_weight, const void *qkv_weight,
                                const void *out_weight, int batch, int row, int NH, int kvNH, int HS, float eps, int dtype)
{
    void *qkv, *att, *output;
    int col = NH * HS;
    int qkv_weight_row = (NH + 2 * kvNH) * HS;
    att = cuda_malloc(batch * row * col * sizeof(float));
    output = cuda_malloc(batch * row * col * sizeof(float));
    qkv = cuda_malloc(batch * row * qkv_weight_row * sizeof(float));

    cuda_rmsnorm_f32(att, embeds, norm_weight, batch * row, col, eps);
    cuda_matmul(qkv, att, qkv_weight, nullptr, batch * row, col, qkv_weight_row, dtype); // (batch * row, col) @ (qkv_weight_row, col)^T
    cuda_rope_qkv(qkv, qkv, freqs, batch, row, NH, kvNH, HS); // rope qkv in-place
    cuda_gq_sdpa(att, qkv, batch, row, NH, kvNH, HS);
    cuda_matmul(output, att, out_weight, nullptr, batch * row, col, col, dtype); // (batch * row, col) @ (col, col)^T
    cuda_add(out, embeds, output, batch * row, col); // residual connect embeddings to attention

    cuda_free(qkv);
    cuda_free(att);
    cuda_free(output);
}

/*
 * Replicate K, V to match the size of Q
 *
 * @param out: output matrix(batch, row, (3 * qNH) * HS)
 * @param inp: input matrix(batch, row, (qNH + 2 * kvNH) * HS) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param qNH: number of Q heads
 * @param kvNH: number of K and V heads
 * @param HS: head size
 */
void cuda_repeat_qkv(void *out, const void *inp, int batch, int row, int qNH, int kvNH, int HS)
{
    const int block_size = 256;
    int total_threads = batch * row * (3 * qNH) * HS; // one thread per output element
    int num_blocks = CEIL_DIV(total_threads, block_size);
    int replicate_factor = qNH / kvNH;
    assert(replicate_factor > 1);
    repeat_qkv_kernel<float><<<num_blocks, block_size, 0, main_stream>>>((float *)out, (const float *)inp, batch, row, qNH, HS, replicate_factor);
}

/*
 * Get the row at the given index
 *
 * @param out: output matrix(batch, col)
 * @param inp: input matrix(batch, row, col)
 * @param batch: batch size
 * @param row: row index
 * @param col: column size
 * @param idx: index. If negative, it is idx from the end.
 */
void cuda_get_row(void *out, const void *inp, int batch, int row, int col, int idx)
{
    int block_size = 8;
    int total_threads = batch;
    int grid_size = CEIL_DIV(total_threads, block_size);

    if (idx < 0)
        idx += row;
    assert(idx >= 0 && idx < row);
    get_row_kernel<<<grid_size, block_size, 0, main_stream>>>((float *)out, (const float *)inp, batch, row, col, idx);
    cuda_check(hipGetLastError());
}

/*
 * Get the idx of the maximum value along the last dimension
 *
 * @param out: output vector(row) as type int
 * @param inp: input matrix(row, col)
 * @param row: row size
 * @param col: column size
 */
void cuda_argmax(void *out, const void *inp, int row, int col)
{
    const int block_size = 256;
    const int grid_size = row;
    argmax_kernel<<<grid_size, block_size, 0, main_stream>>>((int *)out, (const float *)inp, row, col);
    cuda_check(hipGetLastError());
}

void cuda_feed_forward(void *out, const void *attn, const void *norm_weight, const void *fc_weight, const void *out_weight,
                    int batch, int row, int col, int ffl, float eps, int dtype)
{
    void *ffn = cuda_malloc(batch * row * col * sizeof(float));
    void *fc = cuda_malloc(batch * row * 2 * ffl * sizeof(float));

    cuda_rmsnorm_f32(ffn, attn, norm_weight, batch * row, col, eps);
    cuda_matmul(fc, ffn, fc_weight, nullptr, batch * row, col, 2 * ffl, dtype); // (batch * row, col) @ (2 * ffl, col)^T
    cuda_swiglu(fc, fc, batch, row, ffl); // update fc in-place
    cuda_matmul(ffn, fc, out_weight, nullptr, batch * row, ffl, col, dtype); // (batch * row, ffl) @ (col, ffl)^T
    cuda_add(out, attn, ffn, batch * row, col); // residual connect attention to feedforward

    cuda_free(fc);
    cuda_free(ffn);
}

void cuda_classify(void *out, void *ff, const void *norm_weight, const void *out_weight, int batch, int row, int col, int wsize, float eps, int dtype)
{
    void *ffn = (float *)ff + batch * col; // reuse the memory of ff

    assert(batch * 2 <= row);
    cuda_get_row(ff, ff, batch, row, col, -1); // out shape: (batch, col)
    cuda_rmsnorm_f32(ffn, ff, norm_weight, batch, col, eps);
    cuda_matmul(out, ffn, out_weight, nullptr, batch, col, wsize, dtype); // (batch, col) @ (wsize, col)^T
}

void cuda_predict(void *out, void *ff, const void *norm_weight, const void *out_weight, int batch, int row, int col, int wsize, float eps, int dtype)
{
    void *logits = cuda_malloc(batch * wsize * sizeof(float));
    cuda_classify(logits, ff, norm_weight, out_weight, batch, row, col, wsize, eps, dtype);
    cuda_argmax(out, logits, batch, wsize); // TODO: support temp, top_k and top_p
    cuda_free(logits);
}

} // extern "C"
