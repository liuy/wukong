#include "wukong.h"

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
static size_t cublaslt_workspace_size = 32 * 1024 * 1024;
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;
static hipblasLtHandle_t cublaslt_handle;
static hipdnnHandle_t cudnn_handle;
static int cuda_arch_major = 0;
static int cuda_arch_minor = 0;
static int cuda_num_SMs = 0; // for persistent threads where we want 1 threadblock per SM
static int cuda_threads_per_SM = 0;    // needed to calculate how many blocks to launch to fill up the GPU

extern "C" {
void cuda_init(void)
{
    srand(0);   // determinism

    // set up the device
    int deviceIdx = 0;
    cuda_check(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    cuda_num_SMs = deviceProp.multiProcessorCount;
    cuda_threads_per_SM = deviceProp.maxThreadsPerMultiProcessor;
    cuda_arch_major = deviceProp.major;
    cuda_arch_minor = deviceProp.minor;
    printf("CUDA device: %s, major %d, minor %d, num_SMs: %d, threads_per_SM: %d\n",
            deviceProp.name, cuda_arch_major, cuda_arch_minor, cuda_num_SMs, cuda_threads_per_SM);

    cudnn_check(hipdnnCreate(&cudnn_handle));
    cublas_check(hipblasLtCreate(&cublaslt_handle));
    cuda_check(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = cuda_arch_major >= 8 ? 1 : 0;
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
}

void cuda_fini(void)
{
    cuda_check(hipFree(cublaslt_workspace));
    cublas_check(hipblasLtDestroy(cublaslt_handle));
    cudnn_check(hipdnnDestroy(cudnn_handle));
}


void* cuda_malloc(size_t size)
{
    void *ptr;
    cuda_check(hipMalloc(&ptr, size));
    return ptr;
}

void cuda_free(void* ptr)
{
    cuda_check(hipFree(ptr));
}

void cuda_to_device(void* dst, void* src, size_t size)
{
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void cuda_to_host(void* dst, void* src, size_t size)
{
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

/*
 * Fused matrix multiplication with optional bias addition: out = inp @ weight + bias
 *
 * @param out: output matrix(row, oc)
 * @param inp: input matrix(row, column)
 * @param weight: weight matrix(column, oc)
 * @param bias: optional bias vector(oc) (can be NULL)
 * @param row: input row size
 * @param column: input column size
 * @param oc: output column size
 */
void cuda_matmul(void *out, const void *inp, const void *weight, const void *bias,
            int row, int column, int oc)
{
    int res;
    bool has_bias = (bias != nullptr);
    bool has_gelu = false; /* TODO: Fuse GELU */
    hipblasLtMatmulDesc_t desc;
    hipblasLtMatmulPreference_t pref;
    hipblasLtMatrixLayout_t inp_layout, weight_layout, out_layout, bias_layout;
    hipblasLtMatmulHeuristicResult_t heuristic;
    hipblasOperation_t notrans = HIPBLAS_OP_N;
    hipblasLtEpilogue_t epilogue = has_bias ? HIPBLASLT_EPILOGUE_BIAS : HIPBLASLT_EPILOGUE_DEFAULT;

    /*
     * Cuda is colum-major, for row-major Array, if we want to get: out = inp @ weight, 'out' should be 'out.T'.
     * Mathematically, out.T = weight.T @ inp.T. Since cuda is colum-major, 'weight.T' should be weight, 'inp.T' should be inp.
     * so calculating out.T = weight & inp.
     */
    cublas_check(hipblasLtMatmulDescCreate(&desc, cublas_compute_type, HIP_R_32F));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSA, &notrans, sizeof(notrans)));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSB, &notrans, sizeof(notrans)));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    cublas_check(hipblasLtMatrixLayoutCreate(&weight_layout, HIP_R_32F, oc, column, oc));
    cublas_check(hipblasLtMatrixLayoutCreate(&inp_layout, HIP_R_32F, column, row, column));
    cublas_check(hipblasLtMatrixLayoutCreate(&out_layout, HIP_R_32F, oc, row, oc));
    cublas_check(hipblasLtMatrixLayoutCreate(&bias_layout, HIP_R_32F, oc, 1, oc));


    if (has_bias && (uintptr_t)bias % 16 != 0)
        panic("bias must be aligned to 16 bytes");

    cublas_check(hipblasLtMatmulPreferenceCreate(&pref));
    cublas_check(hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    cublas_check(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, desc, weight_layout, inp_layout, out_layout,
                out_layout, pref, 1, &heuristic, &res));
    if (res == 0)
        panic("No algorithm found: row=%d, column=%d, oc=%d, has_bias=%d, has_gelu=%d",
              row, column, oc, has_bias, has_gelu);

    const float alpha = 1.0f, beta = 0.0f;
    cublas_check(hipblasLtMatmul(cublaslt_handle, desc, &alpha, weight, weight_layout, inp, inp_layout, &beta,
                out, out_layout, out, out_layout, &heuristic.algo, cublaslt_workspace, cublaslt_workspace_size, 0));

    cublas_check(hipblasLtMatmulPreferenceDestroy(pref));
    cublas_check(hipblasLtMatmulDescDestroy(desc));
    cublas_check(hipblasLtMatrixLayoutDestroy(weight_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(inp_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(out_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(bias_layout));
}

/*
 * Row-wise cuda_softmax
 * @param output: shape (row, column)
 * @param input: shape (row, column)
 * @param row: row size
 * @param col: column size
 */
void cuda_softmax(void* output, void* input, int row, int col)
{
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    cudnn_check(hipdnnCreateTensorDescriptor(&inputDesc));
    cudnn_check(hipdnnCreateTensorDescriptor(&outputDesc));

    cudnn_check(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, row, col, 1, 1));
    cudnn_check(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, row, col, 1, 1));

    float alpha = 1.0f, beta = 0.0f;
    cudnn_check(hipdnnSoftmaxForward(cudnn_handle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &alpha,
                                    inputDesc, input, &beta, outputDesc, output));

    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
}
}