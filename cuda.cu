#include "hip/hip_runtime.h"
#include "wukong.h"

#define QK_K 256

dtype_info dtype_infos[GGML_TYPE_COUNT] = {
    {"F32", 1, 4},
    {"F16", 1, 2},
    {"GGML_TYPE_Q4_0", 32, 2 + 16},
    {"GGML_TYPE_Q4_1", 32, 2 + 2 + 16},
    {"GGML_TYPE_Q4_2", 32, 2 + 2 + 16},
    {"GGML_TYPE_Q4_3", 32, 2 + 2 + 16},
    {"GGML_TYPE_Q5_0", 32, 2 + 4 + 16},
    {"GGML_TYPE_Q5_1", 32, 2 + 2 + 4 + 16},
    {"GGML_TYPE_Q8_0", 32, 2 + 32},
    {"GGML_TYPE_Q8_1", 32, 4 + 4 + 32},
    {"GGML_TYPE_Q2_K", 256, 2 + 2 + QK_K / 16 + QK_K / 4},
    {"GGML_TYPE_Q3_K", 256, 2 + QK_K / 4 + QK_K / 8 + 12},
    {"GGML_TYPE_Q4_K", 256, 2 + 2 + QK_K / 2 + 12},
    {"GGML_TYPE_Q5_K", 256, 2 + 2 + QK_K / 2 + QK_K / 8 + 12},
    {"GGML_TYPE_Q6_K", 256, 2 + QK_K / 2 + QK_K / 4 + QK_K / 16},
    {"GGML_TYPE_Q8_K", 256, 4 + QK_K + QK_K / 8},
    {"GGML_TYPE_IQ2_XXS", 256, 2 + QK_K / 4},
    {"GGML_TYPE_IQ2_XS", 256, 2 + QK_K / 4 + QK_K / 32},
    {"GGML_TYPE_IQ3_XXS", 256, 2 + QK_K / 4 + QK_K / 8},
    {"GGML_TYPE_IQ1_S", 256, 2 + QK_K / 8 + QK_K / 16},
    {"GGML_TYPE_IQ4_NL", 32, 2 + 16},
    {"GGML_TYPE_IQ3_S", 256, 2 + QK_K / 4 + QK_K / 8 + QK_K / 32 + 4},
    {"GGML_TYPE_IQ2_S", 256, 2 + QK_K / 4 + QK_K / 16},
    {"GGML_TYPE_IQ4_XS", 256, 2 + 2 + QK_K / 2 + QK_K / 64},
    {"Int8", 1, 1},
    {"Int16", 1, 2},
    {"Int32", 1, 4},
    {"Int64", 1, 8},
    {"F64", 1, 8},
    {"GGML_TYPE_IQ1_M", 256, QK_K / 8 + QK_K / 16 + QK_K / 32},
    {"BF16", 1, 2},
    {"GGML_TYPE_Q4_0_4_4", 32, 2 + 16},
    {"GGML_TYPE_Q4_0_4_8", 32, 2 + 16},
    {"GGML_TYPE_Q4_0_8_8", 32, 2 + 16},
    {"GGML_TYPE_TQ1_0", 256, 2 + 4 * 13},
    {"GGML_TYPE_TQ2_0", 256, 2 + 64}
};

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
static size_t cublaslt_workspace_size = 32 * 1024 * 1024;
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;
static hipblasLtHandle_t cublaslt_handle;
static hipblasHandle_t cublas_handle;
static hipdnnHandle_t cudnn_handle;
__attribute_maybe_unused__ static int cuda_arch_major = 0;
__attribute_maybe_unused__ static int cuda_arch_minor = 0;
__attribute_maybe_unused__ static int cuda_num_SMs = 0; // for persistent threads where we want 1 threadblock per SM
__attribute_maybe_unused__ static int cuda_threads_per_SM = 0;    // needed to calculate how many blocks to launch to fill up the GPU
__attribute_maybe_unused__ static int cuda_threads_per_block = 0;
__attribute_maybe_unused__ static int cuda_warp_size = 0; // warp size of the GPU
__attribute_maybe_unused__ static int cuda_max_shared_mem_per_block = 0;

__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__device__ float warp_reduce_max(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

__device__ __forceinline__ void warp_reduce_max(float& val, int& idx) {
    #pragma unroll
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        float other_val = __shfl_down_sync(0xffffffff, val, offset);
        int other_idx = __shfl_down_sync(0xffffffff, idx, offset);
        if (other_val > val) {
            val = other_val;
            idx = other_idx;
        }
    }
}

// Handles both scaling of attention scores and softmax computation with causal masking
// inp/out shape: (B, NH, T, T)
__global__ void scaled_softmax_kernel(float* out, const float* inp, int B, int NH, int T, float scale)
 {
    extern __shared__ float shared[];
    int batch_idx = blockIdx.x / (NH * T); // batch index
    int head_idx = (blockIdx.x / T) % NH;  // head index
    int row_idx = blockIdx.x % T;          // row index within the attention matrix
    int tid = threadIdx.x;
    int warpId = threadIdx.x / WARP_SIZE;         // warp index within a block
    int laneId = threadIdx.x % WARP_SIZE;         // thread index within a warp
    int warpsPerBlock = blockDim.x / WARP_SIZE;

    // shared memory layout: first half for max values, second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // calculate base index for this thread block's row
    int row_start = (batch_idx * NH * T * T) + (head_idx * T * T) + (row_idx * T);
    const float* x = inp + row_start;

    // Step 1: Find maximum while applying scale and causal mask
    float maxval = -INFINITY;
    for (int i = tid; i < T; i += blockDim.x) {
        float val = (i <= row_idx) ? x[i] * scale : -INFINITY;
        maxval = fmaxf(maxval, val);
    }

    // warp-level reduction for maxval
    maxval = warp_reduce_max(maxval);

    // write per-warp maxval to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // final reduction for maxval across warps
    if (tid == 0) {
        float val = maxvals[0];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        maxvals[0] = val;
    }
    __syncthreads();

    // broadcast max to all threads
    float offset = maxvals[0];

    // Step 2: Compute exp(x - max) while respecting causal mask
    float sumval = 0.0f;
    for (int i = tid; i < T; i += blockDim.x) {
        float val = (i <= row_idx) ? expf(x[i] * scale - offset) : 0.0f;
        out[row_start + i] = val;  // store intermediate result
        sumval += val;
    }

    // warp-level reduction for sum
    sumval = warpReduceSum(sumval);

    // write per-warp sum to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // final reduction for sum across warps
    if (tid == 0) {
        float val = sumvals[0];
        for (int i = 1; i < warpsPerBlock; i++) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();

    // Step 3: Normalize by sum
    float sum = sumvals[0];
    float inv_sum = 1.0f / sum;

    // write final normalized values
    for (int i = tid; i < T; i += blockDim.x) {
        if (i <= row_idx) {
            out[row_start + i] *= inv_sum;
        } else {
            out[row_start + i] = 0.0f;
        }
    }
}

__global__ void softmax_kernel(float* output, const float* input, int row, int col) {
    extern __shared__ float shared_mem[];
    float* row_max = shared_mem;                    // First part of shared memory for max values
    float* row_sum = &shared_mem[blockDim.x / WARP_SIZE];  // Second part for sum values

    int tid = threadIdx.x;
    int lane_id = tid % WARP_SIZE;
    int warp_id = tid / WARP_SIZE;
    int warps_per_block = blockDim.x / WARP_SIZE;
    int row_idx = blockIdx.x;

    if (row_idx >= row) return;

    // Step 1: Find maximum value in the row
    float thread_max = -INFINITY;
    for (int i = tid; i < col; i += blockDim.x) {
        thread_max = fmaxf(thread_max, input[row_idx * col + i]);
    }

    // Warp-level reduction for max
    thread_max = warp_reduce_max(thread_max);

    // Store per-warp results
    if (lane_id == 0) {
        row_max[warp_id] = thread_max;
    }
    __syncthreads();

    // Final reduction for max across warps
    if (tid == 0) {
        float max_val = row_max[0];
        for (int i = 1; i < warps_per_block; i++) {
            max_val = fmaxf(max_val, row_max[i]);
        }
        row_max[0] = max_val;
    }
    __syncthreads();

    // Step 2: Compute exp(x - max) and sum
    float max_val = row_max[0];
    float thread_sum = 0.0f;

    for (int i = tid; i < col; i += blockDim.x) {
        float val = expf(input[row_idx * col + i] - max_val);
        output[row_idx * col + i] = val;  // Store intermediate result
        thread_sum += val;
    }

    // Warp-level reduction for sum
    thread_sum = warpReduceSum(thread_sum);

    // Store per-warp sums
    if (lane_id == 0) {
        row_sum[warp_id] = thread_sum;
    }
    __syncthreads();

    // Final reduction for sum across warps
    if (tid == 0) {
        float sum = row_sum[0];
        for (int i = 1; i < warps_per_block; i++) {
            sum += row_sum[i];
        }
        row_sum[0] = sum;
    }
    __syncthreads();

    // Step 3: Normalize by sum
    float inv_sum = 1.0f / row_sum[0];
    for (int i = tid; i < col; i += blockDim.x) {
        output[row_idx * col + i] *= inv_sum;
    }
}

__global__ void unpermute_kernel(float *out, const float * inp, int B, int N, int NH, int d)
{
   // out has shape (B, nh, N, d) but we need to unpermute it to (B, N, nh, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // out[b][n][nh_][d_] <- inp[b][nh_][n][d_]
    if (idx < B * NH * N * d) {
        int b = idx / (NH * N * d);
        int rest = idx % (NH * N * d);
        int nh_ = rest / (N * d);
        rest = rest % (N * d);
        int n = rest / d;
        int d_ = rest % d;

        int other_idx = (b * NH * N * d) + (n * NH * d) + (nh_ * d) + d_;
        out[other_idx] = inp[idx];
    }
}

__global__ void permute_kernel(floatX* q, floatX* k, floatX* v,
                               const floatX* inp,
                               int B, int N, int NH, int d) {
    // okay so now, this kernel wants Q,K,V to all be of shape (B, NH, N, d)
    // but instead, we have a single tensor QKV (inp) of shape (B, N, 3, NH, d)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B * NH * N * d) {
        return;
    }

    // Q[b][nh_][n][d_] = inp[b][n][0][nh_][d_]
    int b = idx / (NH * N * d);
    int rest = idx % (NH * N * d);
    int nh_ = rest / (N * d);
    rest = rest % (N * d);
    int n = rest / d;
    int d_ = rest % d;
    int inp_idx = (b * N * 3 * NH * d) + (n * 3 * NH * d) + (0 * NH * d) + (nh_ * d) + d_;
    q[idx] = __ldcs(&inp[inp_idx]);
    k[idx] = __ldcs(&inp[inp_idx + NH * d]);
    v[idx] = __ldcs(&inp[inp_idx + 2 * (NH * d)]);
}

__global__ void add_bias_kernel(float* out, const float* bias, int T, int OC)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < T * OC; i += stride) {
        int col = i % OC;
        out[i] += bias[col];
    }
}

__global__ void rmsnorm_kernel(float* __restrict__ out, const float* __restrict__ inp,
                              const float* __restrict__ weight, int N, int C, float eps)
{
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> warp = cg::tiled_partition<WARP_SIZE>(block);

    __shared__ float shared_sum2[WARP_SIZE]; // One element per warp for squared sum

    int num_warps = blockDim.x / WARP_SIZE;
    int warp_id = threadIdx.x / WARP_SIZE;
    int lane_id = threadIdx.x % WARP_SIZE;
    int idx = blockIdx.x; // One block per row

    // Point to current sequence position
    const float* x = inp + idx * C;

    // Thread coarsening through the row
    float thread_sum2 = 0.0f;

    // Each thread accumulates multiple elements
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float xi = x[i];
        thread_sum2 += xi * xi;
    }

    // Warp-level reduction for sum of squares
    float warp_sum2 = cg::reduce(warp, thread_sum2, cg::plus<float>{});

    // Store warp-level results to shared memory
    if (lane_id == 0) {
        shared_sum2[warp_id] = warp_sum2;
    }
    __syncthreads();

    // Load results from shared memory to threads, pad with zeros for out-of-bounds threads
    warp_sum2 = (lane_id < num_warps) ? shared_sum2[lane_id] : 0.0f;

    // Reduce the warp-level results
    float block_sum2 = cg::reduce(warp, warp_sum2, cg::plus<float>{});

    block_sum2 /= C; // mean(x**2)
    float s = rsqrtf(block_sum2 + eps); // 1 / sqrt(mean(x**2) + eps)

    // Apply normalization and scaling
    float* o = out + idx * C;
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float val = __ldcs(x + i);
        __stcs(o + i, val * s * weight[i]); // x / sqrt(mean(x**2) + eps) * weight
    }
}

__global__ void swiglu_kernel(floatX* out, const floatX* inp, int B, int T, int C)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < B * T * C) {
        int b = idx / (T * C);
        int t = (idx / C) % T;
        int c = idx % C;

        int fc1_idx = (b * T * 2 * C) + (t * 2 * C) + c;
        int fc2_idx = fc1_idx + C;

        floatX swish_val = inp[fc2_idx] / (1.0f + expf(-inp[fc2_idx]));
        out[idx] = swish_val * inp[fc1_idx];
    }
}

__global__ void rope_qkv_kernel(floatX* out, const floatX* inp, const floatX* raw_freqs,
                                int batch, int row, int NH, int kvNH, int HS)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int HS_half = HS / 2;
    int total_heads = NH + kvNH;
    int total = batch * row * total_heads * HS_half;

    if (idx >= total)
        return;

    int b = idx / (row * total_heads * HS_half);
    int r = (idx / (total_heads * HS_half)) % row;
    int h = (idx / HS_half) % total_heads;
    int d = idx % HS_half;

    float freq = raw_freqs[d];
    float angle = r * freq;
    float c = cosf(angle);
    float s = sinf(angle);

    int base = b * (row * (NH + 2 * kvNH) * HS) + r * ((NH + 2 * kvNH) * HS) + h * HS + 2 * d;
    float x_real = inp[base];
    float x_imag = inp[base + 1];

    out[base]     = x_real * c - x_imag * s;
    out[base + 1] = x_real * s + x_imag * c;
}

__global__ void rope_kernel(floatX *out, const floatX *inp, const floatX *raw_freqs, int B, int T, int NH, int HS)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int HS_half = HS / 2;
    if (idx >= B * T * NH * HS_half)
        return;

    // decode the individual indices
    int b = idx / (T * NH * HS_half);
    int t = (idx / (NH * HS_half)) % T;
    int h = (idx / HS_half) % NH;
    int d = idx % HS_half;
    int idx_bt = b * (T * NH * HS) + t * (NH * HS);
    int idx_bth = idx_bt + h * HS;
    int idxi = idx_bth + 2 * d; // index in the input

    // fetch and compute frequency
    float freq = raw_freqs[d];
    float angle = t * freq;
    float freqs_cos = cosf(angle);
    float freqs_sin = sinf(angle);

    // fetch the input
    float x_real = inp[idxi];
    float x_imag = inp[idxi + 1];
    // apply the rotation
    out[idxi] = x_real * freqs_cos - x_imag * freqs_sin;
    out[idxi + 1] = x_real * freqs_sin + x_imag * freqs_cos;
}

__global__ void get_embeddings_kernel(void* out, const int* inp, const void* embd, int batch, int row, size_t bytes_per_row)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch * row)
        return;

    int b = idx / row;
    int t = idx % row;
    int ix = inp[b * row + t];

    char* dst = (char*)out + (b * row + t) * bytes_per_row;
    const char* src = (const char*)embd + ix * bytes_per_row;

    memcpy(dst, src, bytes_per_row);
}

void cuda_matmul_cublas(float *out, const float *inp, const float *weight, const float *bias,
                        int row, int column, int oc)
{
    const float alpha = 1.0f;
    const float beta = 0.0f;
    // cublas sees us transposed, so we want out(oc, row) = weight(oc, c) @ inp(c, row) + bias
    cublas_check(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, oc, row, column, /* M, N, K*/
                            &alpha, weight, oc, inp, column, &beta, out, oc));
    if (bias != NULL) {
        int block_size = cuda_threads_per_block;
        int grid_size = CEIL_DIV(oc * row, block_size);
        add_bias_kernel<<<grid_size, block_size>>>(out, bias, row, oc);
        cuda_check(hipGetLastError());
    }
}

void cuda_matmul_cublaslt(void *out, const void *inp, const void *weight, const void *bias,
                        int row, int column, int oc)
{
    int res;
    bool has_bias = (bias != nullptr);
    hipblasLtMatmulDesc_t desc;
    hipblasLtMatmulPreference_t pref;
    hipblasLtMatrixLayout_t inp_layout, weight_layout, out_layout, bias_layout;
    hipblasLtMatmulHeuristicResult_t heuristic;
    hipblasOperation_t notrans = HIPBLAS_OP_N;
    hipblasOperation_t trans = HIPBLAS_OP_T;
    hipblasLtEpilogue_t epilogue = has_bias ? HIPBLASLT_EPILOGUE_BIAS : HIPBLASLT_EPILOGUE_DEFAULT;

    /*
     * Cuda is colum-major, for row-major Array, if we want to get: out = inp @ weight.T, 'out' should be 'out.T'.
     * Mathematically, out.T = weight @ inp.T. Since cuda is colum-major, 'weight' should be weight.T, 'inp.T' should be inp.
     * so calculating out.T = weight.T & inp.
     */
    cublas_check(hipblasLtMatmulDescCreate(&desc, cublas_compute_type, HIP_R_32F));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(notrans)));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSB, &notrans, sizeof(notrans)));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    cublas_check(hipblasLtMatrixLayoutCreate(&weight_layout, HIP_R_32F, column, oc, column));
    cublas_check(hipblasLtMatrixLayoutCreate(&inp_layout, HIP_R_32F, column, row, column));
    cublas_check(hipblasLtMatrixLayoutCreate(&out_layout, HIP_R_32F, oc, row, oc));
    cublas_check(hipblasLtMatrixLayoutCreate(&bias_layout, HIP_R_32F, oc, 1, oc));


    if (has_bias && (uintptr_t)bias % 16 != 0)
        panic("bias must be aligned to 16 bytes");

    cublas_check(hipblasLtMatmulPreferenceCreate(&pref));
    cublas_check(hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    cublas_check(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, desc, weight_layout, inp_layout, out_layout,
                out_layout, pref, 1, &heuristic, &res));
    if (res == 0)
        panic("No algorithm found: row=%d, column=%d, oc=%d, has_bias=%d", row, column, oc, has_bias);

    const float alpha = 1.0f, beta = 0.0f;
    cublas_check(hipblasLtMatmul(cublaslt_handle, desc, &alpha, weight, weight_layout, inp, inp_layout, &beta,
                out, out_layout, out, out_layout, &heuristic.algo, cublaslt_workspace, cublaslt_workspace_size, 0));

    cublas_check(hipblasLtMatmulPreferenceDestroy(pref));
    cublas_check(hipblasLtMatmulDescDestroy(desc));
    cublas_check(hipblasLtMatrixLayoutDestroy(weight_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(inp_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(out_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(bias_layout));
}

__global__ void div_kernel(floatX *out, const floatX *a, const floatX *b, int row, int col)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = row * col;

    if (idx < total_elements) {
        out[idx] = a[idx] / b[idx];
    }
}

__global__ void dequantize_Q8_0(float *out, const block_q8_0 *inp, int row, int nb, int bs)
{
    extern __shared__ block_q8_0 shared_block[];
    int block_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_blocks = row * nb;

    if (block_idx >= total_blocks)
        return;

    int r = block_idx / nb; // row index
    int b = block_idx % nb; // block index

    const block_q8_0 *block = inp + r * nb + b;
    shared_block[threadIdx.x] = *block;
    __syncthreads();

    float scale = __half2float(shared_block[threadIdx.x].scale);
    #pragma unroll
    for (int i = 0; i < bs; ++i) {
	    int out_idx = r * nb * bs + b * bs + i;
	    out[out_idx] = scale * shared_block[threadIdx.x].d[i];
    }
}

__global__ void add_kernel(float* out, const float* a, const float* b, int row, int col)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int size = row * col;

    if (idx >= size)
        return;

    float4 *out4 = (float4 *)out;
    const float4 *a4 = (const float4 *)a;
    const float4 *b4 = (const float4 *)b;

    float4 va = a4[idx];
    float4 vb = b4[idx];
    float4 vout;
    vout.x = va.x + vb.x;
    vout.y = va.y + vb.y;
    vout.z = va.z + vb.z;
    vout.w = va.w + vb.w;
    out4[idx] = vout;
}

__global__ void replicate_qkv_kernel(floatX *out, const floatX *inp, int batch, int row, int qNH, int kvNH, int HS)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= batch * row)
        return;

    int b = idx / row;
    int r = idx % row;

    // Base offsets for input and output
    const floatX* inp_row = inp + (b * row * (qNH + 2 * kvNH) * HS) + (r * (qNH + 2 * kvNH) * HS);
    floatX* out_row = out + (b * row * (3 * qNH) * HS) + (r * (3 * qNH) * HS);

    // Copy Q heads
    memcpy(out_row, inp_row, qNH * HS * sizeof(floatX));

    const floatX* k_inp = inp_row + qNH * HS;
    const floatX* v_inp = k_inp + kvNH * HS;

    floatX* k_out = out_row + qNH * HS;
    floatX* v_out = k_out + qNH * HS;

    int gNH = qNH / kvNH;

    // Replicate K heads
    #pragma unroll
    for (int i = 0; i < gNH; i++) {
        memcpy(k_out + i * kvNH * HS, k_inp, kvNH * HS * sizeof(floatX));
    }

    // Replicate V heads
    #pragma unroll
    for (int i = 0; i < gNH; i++) {
        memcpy(v_out + i * kvNH * HS, v_inp, kvNH * HS * sizeof(floatX));
    }
}

__global__ void get_row_kernel(float *out, const float *inp, int batch, int row, int col, int idx)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b < batch) {
        const float *src = inp + b * row * col + idx * col;
        float *dst = out + b * col;
        memcpy(dst, src, col * sizeof(float));
    }
}

__global__ void argmax_kernel(int *out, const float *inp, int row, int col)
{
    __shared__ float smax[WARP_SIZE];  // Max values per warp
    __shared__ int sidx[WARP_SIZE];    // Corresponding indices

    int r = blockIdx.x;                // One row per block
    int tid = threadIdx.x;             // Thread ID
    int wid = tid / WARP_SIZE;         // Warp ID
    int lane = tid % WARP_SIZE;        // Lane within warp

    if (r >= row) return;

    // Each thread's running max
    float max_val = -INFINITY;
    int max_idx = -1;

    for (int i = tid; i < col; i += blockDim.x) {
        float val = inp[r * col + i];
        if (val > max_val) {
            max_val = val;
            max_idx = i;
        }
    }

    // Warp-level reduction
    warp_reduce_max(max_val, max_idx);

    // Write warp results to shared memory
    if (lane == 0) {
        smax[wid] = max_val;
        sidx[wid] = max_idx;
    }
    __syncthreads();

    // Final reduction across warps by first warp
    if (wid == 0) {
        max_val = (lane < blockDim.x/WARP_SIZE) ? smax[lane] : -INFINITY;
        max_idx = (lane < blockDim.x/WARP_SIZE) ? sidx[lane] : -1;

        warp_reduce_max(max_val, max_idx);

        if (lane == 0) {
            out[r] = max_idx;
        }
    }
}

extern "C" {
void cuda_init(void)
{
    srand(0);   // determinism

    // set up the device
    int deviceIdx = 0;
    cuda_check(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    cuda_num_SMs = deviceProp.multiProcessorCount;
    cuda_threads_per_SM = deviceProp.maxThreadsPerMultiProcessor;
    cuda_arch_major = deviceProp.major;
    cuda_arch_minor = deviceProp.minor;
    cuda_threads_per_block = deviceProp.maxThreadsPerBlock;
    cuda_warp_size = deviceProp.warpSize;
    cuda_max_shared_mem_per_block = deviceProp.sharedMemPerBlock;
    // printf("CUDA device: %s, major %d, minor %d, num_SMs: %d, threads_per_SM: %d, threads_per_block: %d, warp_size: %d\n",
    //        deviceProp.name, cuda_arch_major, cuda_arch_minor, cuda_num_SMs, cuda_threads_per_SM, cuda_threads_per_block, cuda_warp_size);

    cudnn_check(hipdnnCreate(&cudnn_handle));
    cublas_check(hipblasCreate(&cublas_handle));
    cublas_check(hipblasLtCreate(&cublaslt_handle));
    cuda_check(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = cuda_arch_major >= 8 ? 1 : 0;
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
}

void cuda_fini(void)
{
    cuda_check(hipFree(cublaslt_workspace));
    cublas_check(hipblasLtDestroy(cublaslt_handle));
    cudnn_check(hipdnnDestroy(cudnn_handle));
}


void* cuda_malloc(size_t size)
{
    void *ptr;
    cuda_check(hipMalloc(&ptr, size));
    return ptr;
}

void cuda_free(void* ptr)
{
    cuda_check(hipFree(ptr));
}

void cuda_to_device(void* dst, void* src, size_t size)
{
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void cuda_to_host(void* dst, void* src, size_t size)
{
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

/*
 * Fused matrix multiplication with optional bias addition: out = inp @ weight^T + bias
 *
 * @param out: output matrix(row, oc)
 * @param inp: input matrix(row, column)
 * @param weight: weight matrix(oc, column)
 * @param bias: optional bias vector(oc) (can be NULL)
 * @param row: input row size
 * @param column: input column size
 * @param oc: output column size
 */
void cuda_matmul(void *out, const void *inp, const void *weight, const void *bias,
                int row, int column, int oc, int dtype)
{
    if (dtype != GGML_TYPE_F32) {
        void *dw = cuda_malloc(oc * column * sizeof(float));
        cuda_dequantize(dw, weight, oc, column, dtype);
        cuda_matmul_cublaslt(out, inp, dw, bias, row, column, oc);
        cuda_free(dw);
        return;
    }
    return cuda_matmul_cublaslt(out, inp, weight, bias, row, column, oc);
}

/*
 * Row-wise cuda_softmax
 * @param output: shape (row, column)
 * @param input: shape (row, column)
 * @param row: row size
 * @param col: column size
 */
void cuda_softmax(void* output, void* input, int row, int col)
{
    const int block_size = 256;
    const int shared_mem_size = (2 * (block_size / WARP_SIZE)) * sizeof(float); // Space for max and sum values
    softmax_kernel<<<row, block_size, shared_mem_size>>>((float *)output, (const float *)input, row, col);
    cuda_check(hipGetLastError());
}

/*
 * GQA scaled dot product attention
 *
 * @param out: output matrix(batch, row, col) where col = qNH * HS
 * @param inp: input matrix(batch, row, (qNH + 2 * kvNH) * HS) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param qNH: number of Q heads
 * @param kvNH: number of K and V heads
 * @param HS: head size
 */
void cuda_gq_sdpa(void *out, const void *inp, int batch, int row, int qNH, int kvNH, int HS)
{
    void *qkv = cuda_malloc(batch * row * 3 * qNH * HS * sizeof(float));
    cuda_replicate_qkv(qkv, inp, batch, row, qNH, kvNH, HS);
    cuda_mh_sdpa(out, qkv, batch, row, qNH, HS);
    cuda_free(qkv);
}

/* Root Mean Square Layer Normalization: x / sqrt(mean(x^2) + eps) * weight
 *
 * @param out: output matrix(row, col)
 * @param inp: input matrix(row, col)
 * @param weight: weight matrix(col)
 * @param row: row size
 * @param col: column size
 * @param eps: epsilon value
 */
void cuda_rmsnorm(void *out, const void *inp, const void *weight, int row, int col, float eps)
{
    const int block_size = 256;
    rmsnorm_kernel<<<row, block_size>>>((floatX *)out, (const floatX *)inp, (const floatX *)weight, row, col, eps);
    cuda_check(hipGetLastError());
}

// swiglu: y = swish(fc2(x)) * fc1(x), where swish(x) = x / (1 + exp(-x)), fc1 and fc2 are fully connected layers
// @param out: output matrix(batch, row, col)
// @param inp: input matrix(batch, row, 2*col), concatenated fc1 and fc2 outputs along the last dimension
void cuda_swiglu(void *out, const void *inp, int batch, int row, int col)
{
    int block_size = 256;
    int grid_size = CEIL_DIV(batch * row * col, block_size);
    swiglu_kernel<<<grid_size, block_size>>>((floatX *)out, (const floatX *)inp, batch, row, col);
    cuda_check(hipGetLastError());
}

/*
 * Vanilla multi-head scaled dot product attention
 *
 * attention = softmax(Q@K^T/sqrt(HS)) @ V
 *
 * @param out: output matrix(batch, row, col)
 * @param inp: input matrix(batch, row, 3 * col) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param NH: number of heads
 * @param HS: head size
 * @attention col = NH * HS
 */
void cuda_mh_sdpa(void *out, const void *inp, int batch, int row, int NH, int HS)
{
    float *qkv, *att, *vatt;

    // Allocate space for broadcasted K and V
    size_t q_size = (batch * NH * row * HS) * sizeof(float);
    size_t qkv_size = 3 * q_size;
    size_t att_size = batch * NH * row * row * sizeof(float);

    cuda_check(hipMalloc(&qkv, qkv_size));
    // try best to reuse input buffer
    vatt = (float *)inp;
    att = vatt + q_size;
    if (att_size > q_size * 2) {
	    cuda_check(hipMalloc(&att, att_size));
    }

    float *q = qkv;
    float *k = qkv + batch * NH * row * HS;
    float *v = k + batch * NH * row * HS;

    // Permute input
    // q: (batch, row, NH, HS) -> (batch, NH, row, HS)
    // k: (batch, row, NH, HS) -> (batch, NH, row, HS)
    // v: (batch, row, NH, HS) -> (batch, NH, row, HS)
    //
    // Tradeoff: it uses more memory for the broadcasted K and V tensors, but this should be acceptable
    // given the benefits in simplicity and performance improvements (remove for-loop in following matmul)
    int total_threads = batch * NH * row * HS;
    int block_size = 256;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    permute_kernel<<<num_blocks, block_size>>>(q, k, v, (const float*)inp, batch, row, NH, HS);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Batched matrix multiplication: Q @ K^T
    cublas_check(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_T, HIPBLAS_OP_N,
                            row, row, HS,
                            &alpha,
                            k, HS, row * HS,
                            q, HS, row * HS,
                            &beta,
                            att, row, row * row,
                            batch * NH));

    // Apply scaled softmax with causal masking
    float scale = 1.0f / sqrtf(HS);
    int softmax_block_size = 256;
    size_t shared_mem_size = 2 * (softmax_block_size / 32) * sizeof(float);
    int grid_size = batch * NH * row;
    scaled_softmax_kernel<<<grid_size, softmax_block_size, shared_mem_size>>>(
        att, att, batch, NH, row, scale);

    // Batched matrix multiplication: attention @ V
    cublas_check(hipblasSgemmStridedBatched(cublas_handle,
                            HIPBLAS_OP_N, HIPBLAS_OP_N,
                            HS, row, row,
                            &alpha,
                            v, HS, row * HS,
                            att, row, row * row,
                            &beta,
                            vatt, HS, row * HS,
                            batch * NH));

    // Unpermute result from (batch, NH, row, HS) -> (batch, row, NH, HS)
    num_blocks = CEIL_DIV(batch * row * NH * HS, block_size);
    unpermute_kernel<<<num_blocks, block_size>>>((float *)out, vatt, batch, row, NH, HS);

    cuda_free(qkv);
    if (att_size > q_size * 2) {
        cuda_free(att);
    }
}

/*
 * Multi query scaled dot product attention
 *
 * @param out: output matrix(batch, row, col) where col = qNH * HS
 * @param inp: input matrix(batch, row, (qNH + 2 * kvNH) * HS) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param qNH: number of Q heads
 * @param HS: head size
 */
void cuda_mq_sdpa(void *out, const void *inp, int batch, int row, int qNH, int HS)
{
    void *qkv = cuda_malloc(batch * row * 3 * qNH * HS * sizeof(float));
    cuda_replicate_qkv(qkv, inp, batch, row, qNH, 1, HS);
    cuda_mh_sdpa(out, qkv, batch, row, qNH, HS);
    cuda_free(qkv);
}

/*
 * RoPE: Rotated Positional Embedding
 *
 * @param out: output matrix(batch, row, NH + 2*kvNH, HS) where NH is for Q and kvNH each for K,V
 * @param inp: input matrix(batch, row, NH + 2*kvNH, HS) q, k, v concatenated along the last dimension
 * @freqs_cis: cos and sin frequencies for each element in q, k
 * @param batch: batch size
 * @param row: row size
 * @param NH: number of query heads
 * @param kvNH: number of key/value heads
 * @param HS: head size
 */
void cuda_rope_qkv(void *out, const void *inp, const void *raw_freqs, int batch, int row, int NH, int kvNH, int HS)
{
    int block_size = 256;
    // We only need threads for Q and K sections, V will be untouched
    int total_threads = batch * row * (NH + kvNH) * HS / 2;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    rope_qkv_kernel<<<num_blocks, block_size>>>((floatX *)out, (const floatX *)inp, (const floatX *)raw_freqs,
                                               batch, row, NH, kvNH, HS);
    cuda_check(hipGetLastError());
}

/*
 * RoPE: Rotated Positional Embedding for a single tensor
 *
 * @param out: output matrix(batch, row, NH, HS)
 * @param inp: input matrix(batch, row, NH, HS)
 * @raw_freqs: raw frequency tensor to compute the rotation angle (HS/2)
 * @param batch: batch size
 * @param row: row size
 * @param NH: number of heads
 * @param HS: head size
 */
void cuda_rope(void *out, const void *inp, const void *raw_freqs, int batch, int row, int NH, int HS)
{
    int block_size = 256;
    int total_threads = batch * row * NH * HS / 2;  // divided by 2 since we process pairs
    int num_blocks = CEIL_DIV(total_threads, block_size);
    rope_kernel<<<num_blocks, block_size>>>((floatX *)out, (const floatX *)inp, (const floatX *)raw_freqs, batch, row, NH, HS);
    cuda_check(hipGetLastError());
}

/*
 * Get the embeddings for the given indices using the embedding table
 *
 * @param out: output matrix(batch, row, col)
 * @param inp: input matrix(batch, row)
 * @param embd: embedding table (vacob_size, col)
 * @param batch: batch size
 * @param row: row size (number of indices)
 * @param col: column size (embedding size)
 */
void cuda_embedding(void* out, const void *inp, const void *embd, int batch, int row, int col, int dtype)
{
    if (dtype < 0 || dtype >= GGML_TYPE_COUNT)
        panic("Unsupported quantization type: %d", dtype);

    auto info = dtype_infos[dtype];
    assert(col % info.block_size == 0);
    size_t bytes_per_row = col / info.block_size * info.type_size;

    const int block_size = 256;
    const int N = batch * row;  // One thread per row
    const int grid_size = CEIL_DIV(N, block_size);

    if (dtype == GGML_TYPE_F32) {
        get_embeddings_kernel<<<grid_size, block_size>>>(out, (const int*)inp, embd, batch, row, bytes_per_row);
        cuda_check(hipGetLastError());
        return;
    }
    void *dout = cuda_malloc(batch * row * bytes_per_row);
    get_embeddings_kernel<<<grid_size, block_size>>>(dout, (const int*)inp, embd, batch, row, bytes_per_row);
    cuda_dequantize(out, dout, batch * row, col, dtype);
    cuda_check(hipGetLastError());
    cuda_free(dout);
}

/*
 * Concatenate the input tensors along the first dimension
 *
 * @param out: output matrix(arow + brow, col)
 * @param a: input matrix(arow, col)
 * @param b: input matrix(brow, col)
 * @param arow: row size of a
 * @param brow: row size of b
 * @param col: column size
 */
void cuda_cat(void *out, const void *a, const void *b, int arow, int brow, int col, int dtype)
{
    auto info = dtype_infos[dtype];
    size_t asize = arow * col * info.type_size / info.block_size;
    size_t bsize = brow * col * info.type_size / info.block_size;

    cuda_check(hipMemcpy(out, a, asize, hipMemcpyDeviceToDevice));
    cuda_check(hipMemcpy((char *)out + asize, b, bsize, hipMemcpyDeviceToDevice));
}

/*
 * Element-wise division a / b
 *
 * @param out: output matrix(row, col)
 * @param a: input matrix(row, col)
 * @param b: input matrix(row, col)
 * @param row: row size
 * @param col: column size
 */
void cuda_div(void *out, const void *a, const void *b, int row, int col)
{
    int block_size = 256;
    int total_threads = row * col;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    div_kernel<<<num_blocks, block_size>>>((floatX *)out, (const floatX *)a, (const floatX *)b, row, col);
    cuda_check(hipGetLastError());
}

/*
 * Dequantize the quantized input tensor from dtype to float
 *
 * @param out: output matrix(row, col)
 * @param inp: input matrix(row, col)
 * @param row: row size
 * @param col: column size
 * @param type: quantization dtype
 */
void cuda_dequantize(void *out, const void *inp, int row, int col, int type)
{
    if (type < 0 || type >= GGML_TYPE_COUNT)
        panic("Unsupported quantization type: %d", type);

    auto info = dtype_infos[type];
    int nb = col / info.block_size;
    int bs = info.block_size;
    int total_blocks = row * nb;
    int block_size = 256;
    int num_blocks = CEIL_DIV(total_blocks, block_size);
    size_t shared_mem_size = block_size * sizeof(block_q8_0);
    assert(shared_mem_size <= cuda_max_shared_mem_per_block);
    switch (type) {
    case GGML_TYPE_Q8_0:
	    dequantize_Q8_0<<<num_blocks, block_size, shared_mem_size>>>((float *)out, (const block_q8_0 *)inp, row, nb, bs);
	    break;
    default:
	    panic("Unsupported quantization type: %s", dtype_infos[type].name);
	}
    cuda_check(hipGetLastError());
}

/*
 * Element-wise addition out = a + b
 *
 * @param out: output matrix(row, col)
 * @param a: input matrix(row, col)
 * @param b: input matrix(row, col)
 * @param row: row size
 * @param col: column size
 */
void cuda_add(void* out, const void* a, const void* b, int row, int col)
{
    const int total_size = row * col;
    const int block_size = 256;
    // Each thread handles 4 elements when using float4
    const int grid_size = CEIL_DIV(total_size, block_size * 4);

    assert(col % 4 == 0);
    add_kernel<<<grid_size, block_size>>>((float*)out, (const float*)a, (const float*)b, row, col);
    cuda_check(hipGetLastError());
}

void cuda_group_query_attention(void *out, const void *embeds, const void *freqs, const void *out_weight, const void *norm_weight,
                                const void *qkv_weight, int batch, int row, int NH, int kvNH, int HS, float eps, int dtype)
{
    void *qkv, *att, *output;
    int col = NH * HS;
    int qkv_weight_row = (NH + 2 * kvNH) * HS;
    att = cuda_malloc(batch * row * col * sizeof(float));
    output = cuda_malloc(batch * row * col * sizeof(float));
    qkv = cuda_malloc(batch * row * qkv_weight_row * sizeof(float));

    cuda_rmsnorm(att, embeds, norm_weight, batch * row, col, eps);
    cuda_matmul(qkv, att, qkv_weight, nullptr, batch * row, col, qkv_weight_row, dtype); // (batch * row, col) @ (qkv_weight_row, col)^T
    cuda_rope_qkv(qkv, qkv, freqs, batch, row, NH, kvNH, HS); // rope qkv in-place
    cuda_gq_sdpa(att, qkv, batch, row, NH, kvNH, HS);
    cuda_matmul(output, att, out_weight, nullptr, batch * row, col, col, dtype); // (batch * row, col) @ (col, col)^T
    cuda_add(out, embeds, output, batch * row, col); // residual connect embeddings to attention

    cuda_free(qkv);
    cuda_free(att);
    cuda_free(output);
}

/*
 * Replicate K, V to match the size of Q
 *
 * @param out: output matrix(batch, row, (3 * qNH) * HS)
 * @param inp: input matrix(batch, row, (qNH + 2 * kvNH) * HS) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param qNH: number of Q heads
 * @param kvNH: number of K and V heads
 * @param HS: head size
 */
void cuda_replicate_qkv(void *out, const void *inp, int batch, int row, int qNH, int kvNH, int HS)
{
    const int block_size = 256;
    int total_threads = batch * row; // copy Q, K, V for each row
    int num_blocks = CEIL_DIV(total_threads, block_size);
    replicate_qkv_kernel<<<num_blocks, block_size>>>((floatX *)out, (const floatX *)inp, batch, row, qNH, kvNH, HS);
}

/*
 * Get the row at the given index
 *
 * @param out: output matrix(batch, col)
 * @param inp: input matrix(batch, row, col)
 * @param batch: batch size
 * @param row: row index
 * @param col: column size
 * @param idx: index. If negative, it is idx from the end.
 */
void cuda_get_row(void *out, const void *inp, int batch, int row, int col, int idx)
{
    int block_size = 8;
    int total_threads = batch;
    int grid_size = CEIL_DIV(total_threads, block_size);

    if (idx < 0)
        idx += row;
    assert(idx >= 0 && idx < row);
    get_row_kernel<<<grid_size, block_size>>>((float *)out, (const float *)inp, batch, row, col, idx);
    cuda_check(hipGetLastError());
}

/*
 * Get the idx of the maximum value along the last dimension
 *
 * @param out: output vector(row)
 * @param inp: input matrix(row, col)
 * @param row: row size
 * @param col: column size
 */
void cuda_argmax(void *out, const void *inp, int row, int col)
{
    const int block_size = 256;
    const int grid_size = row;
    argmax_kernel<<<grid_size, block_size>>>((int *)out, (const float *)inp, row, col);
    cuda_check(hipGetLastError());
}

void cuda_feed_foward(void *out, const void *attn, const void *fc_weight, const void *norm_weight, const void *out_weight,
                    int batch, int row, int col, int ffl, float eps, int dtype)
{
    void *ffn = cuda_malloc(batch * row * col * sizeof(float));
    void *fc = cuda_malloc(batch * row * 2 * ffl * sizeof(float));

    cuda_rmsnorm(ffn, attn, norm_weight, batch * row, col, eps);
    cuda_matmul(fc, ffn, fc_weight, nullptr, batch * row, col, 2 * ffl, dtype); // (batch * row, col) @ (2 * ffl, col)^T
    cuda_swiglu(fc, fc, batch, row, ffl); // update fc in-place
    cuda_matmul(ffn, fc, out_weight, nullptr, batch * row, ffl, col, dtype); // (batch * row, ffl) @ (col, ffl)^T
    cuda_add(out, attn, ffn, batch * row, col); // residual connect attention to feedforward

    cuda_free(fc);
    cuda_free(ffn);
}

void cuda_classify(void *out, void *ff, const void *norm_weight, const void *out_weight, int batch, int row, int col, int wsize, float eps, int dtype)
{
    void *ffn = (float *)ff + batch * col; // reuse the memory of ff

    assert(batch * 2 <= row);
    cuda_get_row(ff, ff, batch, row, col, -1); // out shape: (batch, col)
    cuda_rmsnorm(ffn, ff, norm_weight, batch, col, eps);
    cuda_matmul(out, ffn, out_weight, nullptr, batch, col, wsize, dtype); // (batch, col) @ (wsize, col)^T
}

} // extern "C"
