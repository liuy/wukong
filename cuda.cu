#include "hip/hip_runtime.h"
#include "wukong.h"

// cuBLAS workspace. Hardcoding to 32MiB but only Hopper needs 32, for others 4 is OK
static size_t cublaslt_workspace_size = 32 * 1024 * 1024;
static void* cublaslt_workspace = NULL;
static hipblasComputeType_t cublas_compute_type;
static hipblasLtHandle_t cublaslt_handle;
static hipblasHandle_t cublas_handle;
static hipdnnHandle_t cudnn_handle;
static int cuda_arch_major = 0;
static int cuda_arch_minor = 0;
static int cuda_num_SMs = 0; // for persistent threads where we want 1 threadblock per SM
static int cuda_threads_per_SM = 0;    // needed to calculate how many blocks to launch to fill up the GPU
static int cuda_threads_per_block = 0;
static int cuda_warp_size = 0; // warp size of the GPU

__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

// Handles both scaling of attention scores and softmax computation with causal masking
// inp/out shape: (B, NH, T, T)
__global__ void scaled_softmax_kernel(float* out, const float* inp, int B, int NH, int T, float scale) {
    extern __shared__ float shared[];
    int batch_idx = blockIdx.x / (NH * T); // batch index
    int head_idx = (blockIdx.x / T) % NH;  // head index
    int row_idx = blockIdx.x % T;          // row index within the attention matrix
    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32;         // warp index within a block
    int laneId = threadIdx.x % 32;         // thread index within a warp
    int warpsPerBlock = blockDim.x / 32;

    // shared memory layout: first half for max values, second half for sum values
    float* maxvals = shared;
    float* sumvals = &shared[warpsPerBlock];

    // calculate base index for this thread block's row
    int row_start = (batch_idx * NH * T * T) + (head_idx * T * T) + (row_idx * T);
    const float* x = inp + row_start;

    // Step 1: Find maximum while applying scale and causal mask
    float maxval = -INFINITY;
    for (int i = tid; i < T; i += blockDim.x) {
        float val = (i <= row_idx) ? x[i] * scale : -INFINITY;
        maxval = fmaxf(maxval, val);
    }

    // warp-level reduction for maxval
    maxval = warpReduceMax(maxval);

    // write per-warp maxval to shared memory
    if (laneId == 0) maxvals[warpId] = maxval;
    __syncthreads();

    // final reduction for maxval across warps
    if (tid == 0) {
        float val = maxvals[0];
        for (int i = 1; i < warpsPerBlock; i++) {
            val = fmaxf(val, maxvals[i]);
        }
        maxvals[0] = val;
    }
    __syncthreads();

    // broadcast max to all threads
    float offset = maxvals[0];

    // Step 2: Compute exp(x - max) while respecting causal mask
    float sumval = 0.0f;
    for (int i = tid; i < T; i += blockDim.x) {
        float val = (i <= row_idx) ? expf(x[i] * scale - offset) : 0.0f;
        out[row_start + i] = val;  // store intermediate result
        sumval += val;
    }

    // warp-level reduction for sum
    sumval = warpReduceSum(sumval);

    // write per-warp sum to shared memory
    if (laneId == 0) sumvals[warpId] = sumval;
    __syncthreads();

    // final reduction for sum across warps
    if (tid == 0) {
        float val = sumvals[0];
        for (int i = 1; i < warpsPerBlock; i++) {
            val += sumvals[i];
        }
        sumvals[0] = val;
    }
    __syncthreads();

    // Step 3: Normalize by sum
    float sum = sumvals[0];
    float inv_sum = 1.0f / sum;

    // write final normalized values
    for (int i = tid; i < T; i += blockDim.x) {
        if (i <= row_idx) {
            out[row_start + i] *= inv_sum;
        } else {
            out[row_start + i] = 0.0f;
        }
    }
}

__global__ void softmax_kernel(float* output, const float* input, int row, int col) {
    extern __shared__ float shared_mem[];
    float* row_max = shared_mem;                    // First part of shared memory for max values
    float* row_sum = &shared_mem[blockDim.x / 32];  // Second part for sum values

    int tid = threadIdx.x;
    int lane_id = tid % 32;
    int warp_id = tid / 32;
    int warps_per_block = blockDim.x / 32;
    int row_idx = blockIdx.x;

    if (row_idx >= row) return;

    // Step 1: Find maximum value in the row
    float thread_max = -INFINITY;
    for (int i = tid; i < col; i += blockDim.x) {
        thread_max = fmaxf(thread_max, input[row_idx * col + i]);
    }

    // Warp-level reduction for max
    thread_max = warpReduceMax(thread_max);

    // Store per-warp results
    if (lane_id == 0) {
        row_max[warp_id] = thread_max;
    }
    __syncthreads();

    // Final reduction for max across warps
    if (tid == 0) {
        float max_val = row_max[0];
        for (int i = 1; i < warps_per_block; i++) {
            max_val = fmaxf(max_val, row_max[i]);
        }
        row_max[0] = max_val;
    }
    __syncthreads();

    // Step 2: Compute exp(x - max) and sum
    float max_val = row_max[0];
    float thread_sum = 0.0f;

    for (int i = tid; i < col; i += blockDim.x) {
        float val = expf(input[row_idx * col + i] - max_val);
        output[row_idx * col + i] = val;  // Store intermediate result
        thread_sum += val;
    }

    // Warp-level reduction for sum
    thread_sum = warpReduceSum(thread_sum);

    // Store per-warp sums
    if (lane_id == 0) {
        row_sum[warp_id] = thread_sum;
    }
    __syncthreads();

    // Final reduction for sum across warps
    if (tid == 0) {
        float sum = row_sum[0];
        for (int i = 1; i < warps_per_block; i++) {
            sum += row_sum[i];
        }
        row_sum[0] = sum;
    }
    __syncthreads();

    // Step 3: Normalize by sum
    float inv_sum = 1.0f / row_sum[0];
    for (int i = tid; i < col; i += blockDim.x) {
        output[row_idx * col + i] *= inv_sum;
    }
}

__global__ void gqa_permute_kernel(float* q, float* k, float* v, const float* inp,
                                 int B, int N, int qNH, int kvNH, int HS)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int gNH = qNH / kvNH; // number of query heads per key/value head

    // Handle Q permute: (B, N, qNH, HS) -> (B, gNH, kvNH, N, HS)
    if (idx < B * qNH * N * HS) {
        int b = idx / (qNH * N * HS);
        int rest = idx % (qNH * N * HS);
        int qh = rest / (N * HS);
        rest = rest % (N * HS);
        int n = rest / HS;
        int hs = rest % HS;

        // Calculate group and kv head indices
        int g = qh / kvNH;  // group index
        int kh = qh % kvNH; // corresponding kv head index

        // Input index
        int inp_idx = (b * N * (qNH + 2 * kvNH) * HS) +
                     (n * (qNH + 2 * kvNH) * HS) +
                     (qh * HS) + hs;

        // Output index for Q
        int q_idx = ((b * gNH * kvNH * N * HS) +
                    (g * kvNH * N * HS) +
                    (kh * N * HS) +
                    (n * HS) + hs);

        q[q_idx] = inp[inp_idx];
    }

    // Handle K,V permute: (B, N, kvNH, HS) -> (B, kvNH, N, HS)
    if (idx < B * kvNH * N * HS) {
        int b = idx / (kvNH * N * HS);
        int rest = idx % (kvNH * N * HS);
        int kh = rest / (N * HS);
        rest = rest % (N * HS);
        int n = rest / HS;
        int hs = rest % HS;

        // Input indices
        int k_inp_idx = (b * N * (qNH + 2 * kvNH) * HS) +
                       (n * (qNH + 2 * kvNH) * HS) +
                       (qNH * HS) +
                       (kh * HS) + hs;

        int v_inp_idx = k_inp_idx + (kvNH * HS);

        // Output indices
        int kv_idx = (b * kvNH * N * HS) +
                    (kh * N * HS) +
                    (n * HS) + hs;

        k[kv_idx] = inp[k_inp_idx];
        v[kv_idx] = inp[v_inp_idx];
    }
}

// Transform from (B, gNH, kvNH, N, HS) to (B, N, qNH, HS)
__global__ void gqa_unpermute_kernel(const float* inp, float* out, int B, int N, int qNH, int kvNH, int HS)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int gNH = qNH / kvNH;

    if (idx < B * qNH * N * HS) {
        int b = idx / (qNH * N * HS);
        int rest = idx % (qNH * N * HS);
        int qh = rest / (N * HS);
        rest = rest % (N * HS);
        int n = rest / HS;
        int hs = rest % HS;

        // Calculate group and kv head indices
        int g = qh / kvNH;
        int kh = qh % kvNH;

        // Input index
        int inp_idx = (b * gNH * kvNH * N * HS) +
                     (g * kvNH * N * HS) +
                     (kh * N * HS) +
                     (n * HS) + hs;

        // Output index
        int out_idx = (b * N * qNH * HS) +
                     (n * qNH * HS) +
                     (qh * HS) + hs;

        out[out_idx] = inp[inp_idx];
    }
}

extern "C" {
void cuda_init(void)
{
    srand(0);   // determinism

    // set up the device
    int deviceIdx = 0;
    cuda_check(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    cuda_num_SMs = deviceProp.multiProcessorCount;
    cuda_threads_per_SM = deviceProp.maxThreadsPerMultiProcessor;
    cuda_arch_major = deviceProp.major;
    cuda_arch_minor = deviceProp.minor;
    cuda_threads_per_block = deviceProp.maxThreadsPerBlock;
    cuda_warp_size = deviceProp.warpSize;
    printf("CUDA device: %s, major %d, minor %d, num_SMs: %d, threads_per_SM: %d, threads_per_block: %d, warp_size: %d\n",
           deviceProp.name, cuda_arch_major, cuda_arch_minor, cuda_num_SMs, cuda_threads_per_SM, cuda_threads_per_block, cuda_warp_size);

    cudnn_check(hipdnnCreate(&cudnn_handle));
    cublas_check(hipblasCreate(&cublas_handle));
    cublas_check(hipblasLtCreate(&cublaslt_handle));
    cuda_check(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = cuda_arch_major >= 8 ? 1 : 0;
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
}

void cuda_fini(void)
{
    cuda_check(hipFree(cublaslt_workspace));
    cublas_check(hipblasLtDestroy(cublaslt_handle));
    cudnn_check(hipdnnDestroy(cudnn_handle));
}


void* cuda_malloc(size_t size)
{
    void *ptr;
    cuda_check(hipMalloc(&ptr, size));
    return ptr;
}

void cuda_free(void* ptr)
{
    cuda_check(hipFree(ptr));
}

void cuda_to_device(void* dst, void* src, size_t size)
{
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void cuda_to_host(void* dst, void* src, size_t size)
{
    cuda_check(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

/*
 * Fused matrix multiplication with optional bias addition: out = inp @ weight + bias
 *
 * @param out: output matrix(row, oc)
 * @param inp: input matrix(row, column)
 * @param weight: weight matrix(column, oc)
 * @param bias: optional bias vector(oc) (can be NULL)
 * @param row: input row size
 * @param column: input column size
 * @param oc: output column size
 */
void cuda_matmul(void *out, const void *inp, const void *weight, const void *bias,
            int row, int column, int oc)
{
    int res;
    bool has_bias = (bias != nullptr);
    bool has_gelu = false; /* TODO: Fuse GELU */
    hipblasLtMatmulDesc_t desc;
    hipblasLtMatmulPreference_t pref;
    hipblasLtMatrixLayout_t inp_layout, weight_layout, out_layout, bias_layout;
    hipblasLtMatmulHeuristicResult_t heuristic;
    hipblasOperation_t notrans = HIPBLAS_OP_N;
    hipblasLtEpilogue_t epilogue = has_bias ? HIPBLASLT_EPILOGUE_BIAS : HIPBLASLT_EPILOGUE_DEFAULT;

    /*
     * Cuda is colum-major, for row-major Array, if we want to get: out = inp @ weight, 'out' should be 'out.T'.
     * Mathematically, out.T = weight.T @ inp.T. Since cuda is colum-major, 'weight.T' should be weight, 'inp.T' should be inp.
     * so calculating out.T = weight & inp.
     */
    cublas_check(hipblasLtMatmulDescCreate(&desc, cublas_compute_type, HIP_R_32F));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSA, &notrans, sizeof(notrans)));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSB, &notrans, sizeof(notrans)));
    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));

    cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

    cublas_check(hipblasLtMatrixLayoutCreate(&weight_layout, HIP_R_32F, oc, column, oc));
    cublas_check(hipblasLtMatrixLayoutCreate(&inp_layout, HIP_R_32F, column, row, column));
    cublas_check(hipblasLtMatrixLayoutCreate(&out_layout, HIP_R_32F, oc, row, oc));
    cublas_check(hipblasLtMatrixLayoutCreate(&bias_layout, HIP_R_32F, oc, 1, oc));


    if (has_bias && (uintptr_t)bias % 16 != 0)
        panic("bias must be aligned to 16 bytes");

    cublas_check(hipblasLtMatmulPreferenceCreate(&pref));
    cublas_check(hipblasLtMatmulPreferenceSetAttribute(pref, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

    cublas_check(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, desc, weight_layout, inp_layout, out_layout,
                out_layout, pref, 1, &heuristic, &res));
    if (res == 0)
        panic("No algorithm found: row=%d, column=%d, oc=%d, has_bias=%d, has_gelu=%d",
              row, column, oc, has_bias, has_gelu);

    const float alpha = 1.0f, beta = 0.0f;
    cublas_check(hipblasLtMatmul(cublaslt_handle, desc, &alpha, weight, weight_layout, inp, inp_layout, &beta,
                out, out_layout, out, out_layout, &heuristic.algo, cublaslt_workspace, cublaslt_workspace_size, 0));

    cublas_check(hipblasLtMatmulPreferenceDestroy(pref));
    cublas_check(hipblasLtMatmulDescDestroy(desc));
    cublas_check(hipblasLtMatrixLayoutDestroy(weight_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(inp_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(out_layout));
    cublas_check(hipblasLtMatrixLayoutDestroy(bias_layout));
}

/*
 * Row-wise cuda_softmax
 * @param output: shape (row, column)
 * @param input: shape (row, column)
 * @param row: row size
 * @param col: column size
 */
void cuda_softmax(void* output, void* input, int row, int col)
{
    const int block_size = 256;
    const int shared_mem_size = (2 * (block_size / 32)) * sizeof(float); // Space for max and sum values
    softmax_kernel<<<row, block_size, shared_mem_size>>>((float *)output, (const float *)input, row, col);
    cuda_check(hipGetLastError());
}

/*
 * Vanilla multi-head attention implementation
 *
 * attention = softmax(Q@K^T/sqrt(HS)) @ V
 *
 * @param out: output matrix(batch, row, col)
 * @param inp: input matrix(batch, row, 3 * col) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param NH: number of heads
 * @param HS: head size
 * @attention col = NH * HS
 */
void cuda_mha_attention(void *out, const void *inp, int batch, int row, int NH, int HS)
{
    return cuda_gqa_attention(out, inp, batch, row, NH, NH, HS); // qNH = kvNH
}

/*
 * MQA attention
 * @param out: output matrix(batch, row, col) where col = qNH * HS
 * @param inp: input matrix(batch, row, (qNH + 2 * kvNH) * HS) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param qNH: number of Q heads
 * @param HS: head size
 */
void cuda_mqa_attention(void *out, const void *inp, int batch, int row, int qNH, int HS)
{
    return cuda_gqa_attention(out, inp, batch, row, qNH, 1, HS); // kvNH = 1
}

/*
 * GQA attention
 * @param out: output matrix(batch, row, col) where col = qNH * HS
 * @param inp: input matrix(batch, row, (qNH + 2 * kvNH) * HS) (Q, K, V) concatenated along the last dimension
 * @param batch: batch size
 * @param row: row size
 * @param qNH: number of Q heads
 * @param kvNH: number of K and V heads
 * @param HS: head size
 */
void cuda_gqa_attention(void *out, const void *inp, int batch, int row, int qNH, int kvNH, int HS)
{
    float *workspace, *qkv, *att;
    int gNH = qNH / kvNH;  // number of grouped query heads

    size_t workspace_size = batch * qNH * row * HS * sizeof(float);
    size_t qkv_size = (batch * row * (qNH + 2 * kvNH) * HS) * sizeof(float);
    size_t att_size = batch * gNH * kvNH * row * row * sizeof(float);

    cuda_check(hipMalloc(&workspace, workspace_size));
    cuda_check(hipMalloc(&qkv, qkv_size));
    cuda_check(hipMalloc(&att, att_size));

    float *q = qkv;
    float *k = qkv + batch * row * qNH * HS;
    float *v = k + batch * row * kvNH * HS;

    // 1. Permute and separate input
    // q: (batch, row, qNH, HS) -> (batch, gNH, kvNH, row, HS)
    // k: (batch, row, kvNH, HS) -> (batch, kvNH, row, HS)
    // v: (batch, row, kvNH, HS) -> (batch, kvNH, row, HS)
    int total_threads = batch * qNH * row * HS;
    int block_size = 256;
    int num_blocks = CEIL_DIV(total_threads, block_size);
    gqa_permute_kernel<<<num_blocks, block_size>>>(q, k, v, (const float*)inp,
                                                  batch, row, qNH, kvNH, HS);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // 2. Batched matrix multiplication: Q @ K^T  # (batch, gNH, kvNH, row, HS) @ (batch, kvNH, HS, row) -> (batch, gNH, kvNH, row, row)
    for (int g = 0; g < gNH; g++) {
        float* q_group = q + g * kvNH * batch * row * HS;
        cublas_check(hipblasSgemmStridedBatched(cublas_handle,
                                HIPBLAS_OP_T, HIPBLAS_OP_N,
                                row, row, HS,
                                &alpha,
                                k, HS, row * HS,
                                q_group, HS, row * HS,
                                &beta,
                                att + g * kvNH * batch * row * row,
                                row, row * row,
                                batch * kvNH));
    }

    // 3. Apply scaled softmax with causal masking
    float scale = 1.0f / sqrtf(HS);
    int softmax_block_size = 256;
    size_t shared_mem_size = 2 * (softmax_block_size / 32) * sizeof(float);
    int grid_size = batch * gNH * kvNH * row;
    scaled_softmax_kernel<<<grid_size, softmax_block_size, shared_mem_size>>>(
        att, att, batch * gNH, kvNH, row, scale);

    // 4. Batched matrix multiplication: attention @ V # (batch, gNH, kvNH, row, row) @ (batch, kvNH, row, HS) -> (batch, gNH, kvNH, row, HS)
    for (int g = 0; g < gNH; g++) {
        float* workspace_group = workspace + g * kvNH * batch * row * HS;
        cublas_check(hipblasSgemmStridedBatched(cublas_handle,
                                HIPBLAS_OP_N, HIPBLAS_OP_N,
                                HS, row, row,
                                &alpha,
                                v, HS, row * HS,
                                att + g * kvNH * batch * row * row,
                                row, row * row,
                                &beta,
                                workspace_group, HS, row * HS,
                                batch * kvNH));
    }
    // 5. Unpermute result # (batch, gNH, kvNH, row, HS) -> (batch, row, qNH, HS)
    num_blocks = CEIL_DIV(batch * row * qNH * HS, block_size);
    gqa_unpermute_kernel<<<num_blocks, block_size>>>(workspace, (float *)out, batch, row, qNH, kvNH, HS);

    cuda_check(hipFree(workspace));
    cuda_check(hipFree(qkv));
    cuda_check(hipFree(att));
}

}